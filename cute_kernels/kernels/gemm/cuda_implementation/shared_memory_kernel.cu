#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "include/dtypes.h"
#include "include/math.h"
#include "include/shared_memory.h"
#include "include/threads.h"
#include "index.h"

namespace ck = cute_kernels;

using uint32 = ck::uint32;
using fp32 = ck::fp32;

template <typename scalar_t>
__global__ void _shared_memory_gemm_cuda_kernel(const scalar_t *A,
                                                const scalar_t *B,
                                                const scalar_t *C,
                                                scalar_t *output,
                                                const fp32 alpha,
                                                const fp32 beta,
                                                const uint32 M,
                                                const uint32 K,
                                                const uint32 N) {
    const uint32 i = ck::get_thread_id_along_axis(blockDim.x, blockIdx.y, threadIdx.y);
    const uint32 j = ck::get_thread_id_along_axis(blockDim.x, blockIdx.x, threadIdx.x);

    scalar_t *shared_memory = ck::get_dynamic_shared_memory<scalar_t>();

    scalar_t *A_shared = shared_memory;
    scalar_t *B_shared = &shared_memory[blockDim.x * blockDim.x];

    fp32 accumulator = 0;

    // clang-format off
    #pragma unroll 128
    // clang-format on
    for (uint32 k = 0; k < K; k += blockDim.x) {
        const uint32 index = get_matrix_index<uint32>(threadIdx.y, threadIdx.x, blockDim.x, blockDim.x, false);

        // instead of looping over k dimension, we use the threads in the block to load the data to shared memory
        uint32 k_offset = k + threadIdx.x;
        if (i < M && k_offset < K) {
            A_shared[index] = A[get_matrix_index<uint32>(i, k_offset, M, K, false)];
        }

        // instead of looping over k dimension, we use the threads in the block to load the data to shared memory
        k_offset = k + threadIdx.y;
        if (j < N && k_offset < K) {
            B_shared[index] = B[get_matrix_index<uint32>(k_offset, j, K, N, false)];
        }

        __syncthreads();

        if (i < M && j < N) {
            const uint32 max_q = min(K - k, blockDim.x);
            for (uint32 q = 0; q < max_q; q++) {
                accumulator += A_shared[get_matrix_index<uint32>(threadIdx.y, q, blockDim.x, blockDim.x, false)] *
                               B_shared[get_matrix_index<uint32>(q, threadIdx.x, blockDim.x, blockDim.x, false)];
            }
        }

        // needed for ensuring that shared memory buffers are not modified before the loop finishes for all threads
        __syncthreads();
    }

    if (i < M && j < N) {
        accumulator *= alpha;
        const uint32 index = get_matrix_index<uint32>(i, j, M, N, false);

        if (beta != 0) {
            accumulator += beta * C[index];
        }

        output[index] = accumulator;
    }
}

void shared_memory_gemm_cuda(const torch::Tensor &A,
                             const torch::Tensor &B,
                             std::optional<torch::Tensor> &C,
                             torch::Tensor &output,
                             const bool &is_A_transposed,
                             const bool &is_B_transposed,
                             const fp32 &alpha,
                             const fp32 &beta,
                             const uint32 &M,
                             const uint32 &K,
                             const uint32 &N,
                             const uint32 &BLOCK_SIZE) {
    TORCH_CHECK((BLOCK_SIZE * BLOCK_SIZE) % WARP_SIZE == 0);

    TORCH_CHECK(!is_A_transposed);
    TORCH_CHECK(!is_B_transposed);

    dim3 NUM_BLOCKS = dim3(ck::ceil_divide<uint32>(N, BLOCK_SIZE), ck::ceil_divide<uint32>(M, BLOCK_SIZE), 1);
    dim3 BLOCK_SIZE_dim = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

    AT_DISPATCH_CUSTOM_FLOAT_TYPES(
        A.scalar_type(), "shared_memory_gemm_cuda_kernel", ([&] {
            _shared_memory_gemm_cuda_kernel<scalar_t>
                <<<NUM_BLOCKS, BLOCK_SIZE_dim, 2 * BLOCK_SIZE * BLOCK_SIZE * sizeof(scalar_t)>>>(
                    A.data_ptr<scalar_t>(),
                    B.data_ptr<scalar_t>(),
                    C.has_value() ? C.value().data_ptr<scalar_t>() : nullptr,
                    output.data_ptr<scalar_t>(),
                    alpha,
                    beta,
                    M,
                    K,
                    N);
        }));
}
