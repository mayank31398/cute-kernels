#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "../../../include/dtypes/all.h"
#include "../../../include/math.h"
#include "../../../include/threads.h"
#include "index.h"

template <typename scalar_t>
__global__ void _shared_memory_gemm_cuda_kernel(const scalar_t *a,
                                                const scalar_t *b,
                                                const scalar_t *c,
                                                scalar_t *output,
                                                const fp32 alpha,
                                                const fp32 beta,
                                                const uint32 M,
                                                const uint32 K,
                                                const uint32 N) {
    const uint32 i = get_thread_id_along_axis(blockDim.x, blockIdx.y, threadIdx.y);
    const uint32 j = get_thread_id_along_axis(blockDim.x, blockIdx.x, threadIdx.x);

    if (i < M && j < N) {
        fp32 accumulator = 0;
        uint32 k;

        for (k = 0; k < K; k += blockDim.x) {
            __shared__ scalar_t a_shared[blockDim.x][blockDim.x];
            __shared__ scalar_t b_shared[blockDim.x][blockDim.x];

            if (k < K) {
                a_shared[threadIdx.y][threadIdx.x] = a[get_matrix_index(i, k, M, K, false)];
                b_shared[threadIdx.y][threadIdx.x] = b[get_matrix_index(k, j, K, N, false)];
            }

            __syncthreads();

            const uint32 max_q = min(K - k, blockDim.x);
            for (uint32 q = 0; q < max_q; q++) {
                accumulator += a_shared[i][q] * b_shared[q][j];
            }

            __syncthreads();
        }

        accumulator *= alpha;
        const uint64 index = get_matrix_index(i, j, M, N, false);

        if (beta != 0) {
            accumulator += beta * c[index];
        }

        output[index] = accumulator;
    }
}

void shared_memory_gemm_cuda(const torch::Tensor &a,
                             const torch::Tensor &b,
                             std::optional<torch::Tensor> &c,
                             torch::Tensor &output,
                             const bool &is_a_transposed,
                             const bool &is_b_transposed,
                             const fp32 alpha,
                             const fp32 beta,
                             const uint32 &M,
                             const uint32 &K,
                             const uint32 &N,
                             const uint32 &BLOCK_SIZE) {
    TORCH_CHECK((BLOCK_SIZE * BLOCK_SIZE) % WARP_SIZE == 0);

    TORCH_CHECK(!is_a_transposed);
    TORCH_CHECK(!is_b_transposed);

    dim3 NUM_BLOCKS = dim3(ceil_divide<uint32>(N, BLOCK_SIZE), ceil_divide<uint32>(M, BLOCK_SIZE), 1);

    AT_DISPATCH_CUSTOM_FLOAT_TYPES(a.scalar_type(), "shared_memory_gemm_cuda_kernel", ([&] {
                                       _shared_memory_gemm_cuda_kernel<scalar_t><<<NUM_BLOCKS, BLOCK_SIZE>>>(
                                           a.data_ptr<scalar_t>(),
                                           b.data_ptr<scalar_t>(),
                                           c.has_value() ? c.value().data_ptr<scalar_t>() : nullptr,
                                           output.data_ptr<scalar_t>(),
                                           alpha,
                                           beta,
                                           M,
                                           K,
                                           N);
                                   }));
}
