#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "include/cute_kernels.h"
#include "index.cuh"

namespace ck = cute_kernels;
namespace ck_mem = cute_kernels::memory;

using uint32 = ck::uint32;
using fp32 = ck::fp32;

template <typename scalar_t>
__global__ void _shared_memory_gemm_cuda_kernel(const scalar_t *A,
                                                const scalar_t *B,
                                                const scalar_t *C,
                                                scalar_t *output,
                                                const fp32 alpha,
                                                const fp32 beta,
                                                const uint32 M,
                                                const uint32 K,
                                                const uint32 N) {
    const uint32 i = blockIdx.y * blockDim.x + threadIdx.y;
    const uint32 j = blockIdx.x * blockDim.x + threadIdx.x;

    scalar_t *shared_memory = ck_mem::get_dynamic_shared_memory<scalar_t>();

    scalar_t *A_shared = shared_memory;
    scalar_t *B_shared = &shared_memory[blockDim.x * blockDim.x];

    fp32 accumulator = 0;

    // clang-format off
    #pragma unroll 128
    // clang-format on
    for (uint32 k = 0; k < K; k += blockDim.x) {
        const uint32 index = get_matrix_index<uint32, false>(threadIdx.y, threadIdx.x, blockDim.x, blockDim.x);

        // instead of looping over k dimension, we use the threads in the block to load the data to shared memory
        uint32 k_offset = k + threadIdx.x;
        if (i < M && k_offset < K) {
            A_shared[index] = A[get_matrix_index<uint32, false>(i, k_offset, M, K)];
        }

        // instead of looping over k dimension, we use the threads in the block to load the data to shared memory
        k_offset = k + threadIdx.y;
        if (j < N && k_offset < K) {
            B_shared[index] = B[get_matrix_index<uint32, false>(k_offset, j, K, N)];
        }

        __syncthreads();

        if (i < M && j < N) {
            const uint32 max_q = min(K - k, blockDim.x);
            for (uint32 q = 0; q < max_q; q++) {
                accumulator += A_shared[get_matrix_index<uint32, false>(threadIdx.y, q, blockDim.x, blockDim.x)] *
                               B_shared[get_matrix_index<uint32, false>(q, threadIdx.x, blockDim.x, blockDim.x)];
            }
        }

        // needed for ensuring that shared memory buffers are not modified before the loop finishes for all threads
        __syncthreads();
    }

    if (i < M && j < N) {
        accumulator *= alpha;
        const uint32 index = get_matrix_index<uint32, false>(i, j, M, N);

        if (beta != 0) {
            accumulator += beta * C[index];
        }

        output[index] = accumulator;
    }
}

void shared_memory_gemm_cuda(const torch::Tensor &A,
                             const torch::Tensor &B,
                             std::optional<torch::Tensor> &C,
                             torch::Tensor &output,
                             const bool &is_A_transposed,
                             const bool &is_B_transposed,
                             const fp32 &alpha,
                             const fp32 &beta,
                             const uint32 &M,
                             const uint32 &K,
                             const uint32 &N,
                             const uint32 &BLOCK_SIZE) {
    CHECK_CUDA_TENSOR(A);
    CHECK_CUDA_TENSOR(B);
    CHECK_CUDA_TENSOR(output);

    if (C.has_value()) {
        CHECK_CUDA_TENSOR(C.value());
    }

    CHECK_VALID_THREAD_BLOCK(BLOCK_SIZE);

    TORCH_CHECK(!is_A_transposed);
    TORCH_CHECK(!is_B_transposed);

    dim3 NUM_BLOCKS = dim3(ck::ceil_divide<uint32>(N, BLOCK_SIZE), ck::ceil_divide<uint32>(M, BLOCK_SIZE), 1);
    dim3 BLOCK_SIZE_dim = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

    DISPATCH_FLOAT_KERNEL(A.scalar_type(), "shared_memory_gemm_cuda_kernel", scalar_t, ([&] {
                              _shared_memory_gemm_cuda_kernel<scalar_t>
                                  <<<NUM_BLOCKS, BLOCK_SIZE_dim, 2 * BLOCK_SIZE * BLOCK_SIZE * sizeof(scalar_t)>>>(
                                      A.data_ptr<scalar_t>(),
                                      B.data_ptr<scalar_t>(),
                                      C.has_value() ? C.value().data_ptr<scalar_t>() : nullptr,
                                      output.data_ptr<scalar_t>(),
                                      alpha,
                                      beta,
                                      M,
                                      K,
                                      N);
                          }));
}
