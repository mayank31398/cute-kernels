#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "../../../../include/dtypes/all.h"
#include "../../../../include/threads.h"

template <typename scalar_t, typename vector_t>
__global__ void _add_tensor_forward_cuda_kernel(const scalar_t *x,
                                                const scalar_t *y,
                                                scalar_t *output,
                                                const int64_t num_elements) {
    constexpr int vector_instruction_width = sizeof(vector_t) / sizeof(scalar_t);
    static_assert(vector_instruction_width == 1 || vector_instruction_width == 2 || vector_instruction_width == 4 ||
                  vector_instruction_width == 8 || vector_instruction_width == 16);

    using dtype = DType<scalar_t>;
    using T = typename dtype::nv_dtype;
    using T2 = typename dtype::nv_dtype2;

    const uint64 thread_id = get_global_thread_id();

    if constexpr (vector_instruction_width == 1) {
        if (thread_id < num_elements) {
            output[thread_id] = x[thread_id] + y[thread_id];
        }
    } else {
        uint64 end = (thread_id + 1) * vector_instruction_width - 1;  // inclusive of last element

        if (end < num_elements) {
            vector_t *output_vec = (vector_t *)output;

            if constexpr (std::is_same_v<scalar_t, fp32>) {
                if constexpr (vector_instruction_width == 8) {
                    const fp64 *x_vec = (fp64 *)&((vector_t *)x)[thread_id];
                    const fp64 *y_vec = (fp64 *)&((vector_t *)y)[thread_id];

                    constexpr int n = vector_instruction_width >> 1;
                    fp64 output_buffer[n];

                    // clang-format off
                    #pragma unroll
                    // clang-format on
                    for (int i = 0; i < n; i++) {
                        T2 _x = dtype::reinterpret_64_bits_as_2x32(x_vec[i]);
                        T2 _y = dtype::reinterpret_64_bits_as_2x32(y_vec[i]);

                        output_buffer[i] = dtype::reinterpret_2x32_as_64_bits(_x.x + _y.x, _x.y + _y.y);
                    }

                    output_vec[thread_id] = DType<fp64>::make4(output_buffer);
                } else {
                    const fp32 *x_vec = (fp32 *)&((vector_t *)x)[thread_id];
                    const fp32 *y_vec = (fp32 *)&((vector_t *)y)[thread_id];
                    fp32 output_buffer[vector_instruction_width];

                    // clang-format off
                    #pragma unroll
                    // clang-format on
                    for (int i = 0; i < vector_instruction_width; i++) {
                        output_buffer[i] = x_vec[i] + y_vec[i];
                    }

                    if constexpr (vector_instruction_width == 2) {
                        output_vec[thread_id] = dtype::make2(output_buffer);
                    } else if constexpr (vector_instruction_width == 4) {
                        output_vec[thread_id] = dtype::make4(output_buffer);
                    } else {
                        static_assert("vector_instruction_width is invalid for fp32");
                    }
                }
            } else {
                if constexpr (vector_instruction_width == 2) {
                    const T2 _x = ((vector_t *)x)[thread_id];
                    const T2 _y = ((vector_t *)y)[thread_id];

                    output_vec[thread_id] = __hadd2(_x, _y);
                } else if (vector_instruction_width == 16) {
                    const fp64 *x_vec = (fp64 *)&((vector_t *)x)[thread_id];
                    const fp64 *y_vec = (fp64 *)&((vector_t *)y)[thread_id];

                    constexpr int n = vector_instruction_width >> 2;
                    fp64 output_buffer[n];

                    // clang-format off
                    #pragma unroll
                    // clang-format on
                    for (int i = 0; i < n; i++) {
                        auto [x_first, x_second, x_third, x_fourth] = dtype::reinterpret_64_bits_as_4x16(x_vec[i]);
                        auto [y_first, y_second, y_third, y_fourth] = dtype::reinterpret_64_bits_as_4x16(y_vec[i]);

                        T2 x_left = dtype::make2(x_first, x_second);
                        T2 y_left = dtype::make2(y_first, y_second);
                        x_left = __hadd2(x_left, y_left);

                        T2 x_right = dtype::make2(x_third, x_fourth);
                        T2 y_right = dtype::make2(y_third, y_fourth);
                        x_right = __hadd2(x_right, y_right);

                        output_buffer[i] = dtype::reinterpret_4x16_as_64_bits(x_left, x_right);
                    }

                    output_vec[thread_id] = DType<fp64>::make4(output_buffer);
                } else {
                    const fp32 *x_vec = (fp32 *)&((vector_t *)x)[thread_id];
                    const fp32 *y_vec = (fp32 *)&((vector_t *)y)[thread_id];

                    constexpr int n = vector_instruction_width >> 1;
                    fp32 output_buffer[n];

                    // clang-format off
                    #pragma unroll
                    // clang-format on
                    for (int i = 0; i < n; i++) {
                        T2 _x = dtype::reinterpret_32_bits_as_2x16(x_vec[i]);
                        T2 _y = dtype::reinterpret_32_bits_as_2x16(y_vec[i]);

                        _x = __hadd2(_x, _y);
                        output_buffer[i] = dtype::reinterpret_2x16_as_32_bits(_x);
                    }

                    if constexpr (vector_instruction_width == 4) {
                        output_vec[thread_id] = DType<fp32>::make2(output_buffer);
                    } else if constexpr (vector_instruction_width == 8) {
                        output_vec[thread_id] = DType<fp32>::make4(output_buffer);
                    } else {
                        static_assert("vector_instruction_width is invalid for fp16 & bf16");
                    }
                }
            }
        }

        // use first warp for computing the last elements
        if (thread_id < WARP_SIZE) {
            // NOTE end is same as start since we don't use vector load stores here
            end = (num_elements / vector_instruction_width) * vector_instruction_width + thread_id;
            if (end < num_elements) {
                output[end] = x[end] + y[end];
            }
        }
    }
}

void add_tensor_forward_cuda(const torch::Tensor &x,
                             const torch::Tensor &y,
                             torch::Tensor &output,
                             const int &vector_instruction_width,
                             const int &BLOCK_SIZE) {
    assert(BLOCK_SIZE % WARP_SIZE == 0);

    const int64_t num_elements = x.numel();

    const int num_elements_per_block = BLOCK_SIZE * vector_instruction_width;
    const int NUM_BLOCKS = (num_elements + num_elements_per_block - 1) / num_elements_per_block;

    AT_DISPATCH_CUSTOM_FLOAT_TYPES(
        x.scalar_type(), "add_tensor_forward_cuda_kernel", ([&] {
            switch (vector_instruction_width) {
                case 1:
                    _add_tensor_forward_cuda_kernel<scalar_t, scalar_t><<<NUM_BLOCKS, BLOCK_SIZE>>>(
                        x.data_ptr<scalar_t>(), y.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(), num_elements);
                    break;
                case 2:
                    using vector_t = typename DType<scalar_t>::nv_dtype2;
                    _add_tensor_forward_cuda_kernel<scalar_t, vector_t><<<NUM_BLOCKS, BLOCK_SIZE>>>(
                        x.data_ptr<scalar_t>(), y.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(), num_elements);
                    break;
                case 4:
                    if constexpr (std::is_same_v<scalar_t, fp32>) {
                        _add_tensor_forward_cuda_kernel<scalar_t, fp32_4><<<NUM_BLOCKS, BLOCK_SIZE>>>(
                            x.data_ptr<scalar_t>(), y.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(), num_elements);
                    } else {
                        _add_tensor_forward_cuda_kernel<scalar_t, fp32_2><<<NUM_BLOCKS, BLOCK_SIZE>>>(
                            x.data_ptr<scalar_t>(), y.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(), num_elements);
                    }
                    break;
                case 8:
                    if constexpr (std::is_same_v<scalar_t, fp32>) {
                        _add_tensor_forward_cuda_kernel<scalar_t, fp64_4><<<NUM_BLOCKS, BLOCK_SIZE>>>(
                            x.data_ptr<scalar_t>(), y.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(), num_elements);
                    } else {
                        _add_tensor_forward_cuda_kernel<scalar_t, fp32_4><<<NUM_BLOCKS, BLOCK_SIZE>>>(
                            x.data_ptr<scalar_t>(), y.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(), num_elements);
                    }
                    break;
                case 16:
                    if constexpr (std::is_same_v<scalar_t, c10::Half> || std::is_same_v<scalar_t, c10::BFloat16>) {
                        _add_tensor_forward_cuda_kernel<scalar_t, fp64_4><<<NUM_BLOCKS, BLOCK_SIZE>>>(
                            x.data_ptr<scalar_t>(), y.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(), num_elements);
                    } else {
                        throw std::runtime_error("invalid vector_instruction_width = 16 for fp32");
                    }
                    break;
                default:
                    throw std::runtime_error("invalid vector_instruction_width");
                    break;
            }
        }));
}
