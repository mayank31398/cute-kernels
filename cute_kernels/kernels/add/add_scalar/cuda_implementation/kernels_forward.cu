#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "../../../../include/dtypes/all.h"
#include "../../../../include/launch.h"
#include "../../../../include/math.h"
#include "../../../../include/threads.h"

template <typename scalar_t, typename vector_t>
__global__ void _add_scalar_forward_cuda_kernel(const scalar_t *x,
                                                const fp32 y,
                                                scalar_t *output,
                                                const uint num_elements) {
    constexpr int vector_instruction_width = sizeof(vector_t) / sizeof(scalar_t);
    static_assert(vector_instruction_width == 1 || vector_instruction_width == 2 || vector_instruction_width == 4 ||
                  vector_instruction_width == 8);

    using dtype = DType<scalar_t>;
    using T = typename dtype::nv_dtype;
    using T2 = typename dtype::nv_dtype2;

    const uint thread_id = get_global_thread_id();

    if constexpr (vector_instruction_width == 1) {
        if (thread_id < num_elements) {
            output[thread_id] = x[thread_id] + y;
        }
    } else {
        uint end = (thread_id + 1) * vector_instruction_width - 1;  // inclusive of last element

        if (end < num_elements) {
            vector_t *output_vec = (vector_t *)output;

            if constexpr (std::is_same_v<scalar_t, fp32>) {
                const fp32 *x_vec = (fp32 *)&((vector_t *)x)[thread_id];
                fp32 output_buffer[vector_instruction_width];

                // clang-format off
                #pragma unroll
                // clang-format on
                for (int i = 0; i < vector_instruction_width; i++) {
                    output_buffer[i] = x_vec[i] + y;
                }

                if constexpr (vector_instruction_width == 2) {
                    output_vec[thread_id] = dtype::make2(output_buffer);
                } else if constexpr (vector_instruction_width == 4) {
                    output_vec[thread_id] = dtype::make4(output_buffer);
                } else {
                    static_assert("vector_instruction_width is invalid for fp32");
                }
            } else {
                if constexpr (vector_instruction_width == 2) {
                    const T2 _x = ((vector_t *)x)[thread_id];
                    fp32_2 _x_upcast = dtype::upcast(_x);

                    _x_upcast = DType<fp32>::make2(_x_upcast.x + y, _x_upcast.y + y);
                    output_vec[thread_id] = dtype::downcast(_x_upcast);
                } else {
                    const fp32 *x_vec = (fp32 *)&((vector_t *)x)[thread_id];

                    constexpr int n = vector_instruction_width >> 1;
                    fp32 output_buffer[n];

                    // clang-format off
                    #pragma unroll
                    // clang-format on
                    for (int i = 0; i < n; i++) {
                        fp32_2 _x_upcast = dtype::upcast(dtype::reinterpret_32_bits_as_2x16(x_vec[i]));
                        _x_upcast = DType<fp32>::make2(_x_upcast.x + y, _x_upcast.y + y);
                        output_buffer[i] = dtype::reinterpret_2x16_as_32_bits(dtype::downcast(_x_upcast));
                    }

                    if constexpr (vector_instruction_width == 4) {
                        output_vec[thread_id] = DType<fp32>::make2(output_buffer);
                    } else if constexpr (vector_instruction_width == 8) {
                        output_vec[thread_id] = DType<fp32>::make4(output_buffer);
                    } else {
                        static_assert("vector_instruction_width is invalid for fp16 & bf16");
                    }
                }
            }
        }

        // use first warp for computing the last elements
        if (thread_id < WARP_SIZE) {
            // NOTE end is same as start since we don't use vector load stores here
            end = (num_elements / vector_instruction_width) * vector_instruction_width + thread_id;
            if (end < num_elements) {
                output[end] = x[end] + y;
            }
        }
    }
}

void add_scalar_forward_cuda(const torch::Tensor &x,
                             const float &y,
                             torch::Tensor &output,
                             const int &vector_instruction_width,
                             const int &BLOCK_SIZE) {
    assert(BLOCK_SIZE % WARP_SIZE == 0);
    const uint64 total_elements = x.numel();

    AT_DISPATCH_CUSTOM_FLOAT_TYPES(
        x.scalar_type(), "add_scalar_forward_cuda_kernel", ([&] {
            std::vector<ChunkedArray<scalar_t>> x_chunked =
                chunk_array<scalar_t>(x.data_ptr<scalar_t>(), total_elements);
            std::vector<ChunkedArray<scalar_t>> output_chunked =
                chunk_array<scalar_t>(output.data_ptr<scalar_t>(), total_elements);

            const uint num_elements = x_chunked.num_elements;

            scalar_t *x_chunk = x_chunked.array;
            scalar_t *output_chunk = output_chunked.array;

            const uint num_elements_per_block = BLOCK_SIZE * vector_instruction_width;
            const uint NUM_BLOCKS = ceil_divide<uint>(num_elements, num_elements_per_block);

            for (int i = 0; i < x_chunked.size(); i++) {
                ChunkedArray<scalar_t> x_chunk = x_chunked[i];
                ChunkedArray<scalar_t> output_chunk = output_chunked[i];

                switch (vector_instruction_width) {
                    case 1:
                        _add_scalar_forward_cuda_kernel<scalar_t, scalar_t>
                            <<<NUM_BLOCKS, BLOCK_SIZE>>>(x_chunk.array, y, output_chunk.array, x_chunk.num_elements);
                        break;
                    case 2:
                        using vector_t = typename DType<scalar_t>::nv_dtype2;
                        _add_scalar_forward_cuda_kernel<scalar_t, vector_t>
                            <<<NUM_BLOCKS, BLOCK_SIZE>>>(x_chunk.array, y, output_chunk.array, x_chunk.num_elements);
                        break;
                    case 4:
                        if constexpr (std::is_same_v<scalar_t, fp32>) {
                            _add_scalar_forward_cuda_kernel<scalar_t, fp32_4><<<NUM_BLOCKS, BLOCK_SIZE>>>(
                                x_chunk.array, y, output_chunk.array, x_chunk.num_elements);
                        } else {
                            _add_scalar_forward_cuda_kernel<scalar_t, fp32_2><<<NUM_BLOCKS, BLOCK_SIZE>>>(
                                x_chunk.array, y, output_chunk.array, x_chunk.num_elements);
                        }
                        break;
                    case 8:
                        if constexpr (std::is_same_v<scalar_t, fp32>) {
                            _add_scalar_forward_cuda_kernel<scalar_t, fp64_4><<<NUM_BLOCKS, BLOCK_SIZE>>>(
                                x_chunk.array, y, output_chunk.array, x_chunk.num_elements);
                        } else {
                            _add_scalar_forward_cuda_kernel<scalar_t, fp32_4><<<NUM_BLOCKS, BLOCK_SIZE>>>(
                                x_chunk.array, y, output_chunk.array, x_chunk.num_elements);
                        }
                        break;
                    default:
                        throw std::runtime_error("invalid vector_instruction_width");
                        break;
                }
            }
        }));
}
