#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "include/cute_kernels.h"

namespace ck = cute_kernels;
namespace ck_mem = ck::memory;

using fp32 = ck::fp32;
using fp32_2 = ck::fp32_2;
using fp32_4 = ck::fp32_4;

using uint32 = ck::uint32;
using uint64 = ck::uint64;

template <typename scalar_t>
__global__ void _add_scalar_cuda_kernel(const scalar_t *x, const fp32 y, scalar_t *output, const uint64 num_elements) {
    constexpr uint32 num_elements_per_thread = ck_mem::Packed128<scalar_t>::size;

    const uint32 thread_id = ck::get_global_thread_id();
    const uint32 num_vector_elements = num_elements / num_elements_per_thread;

    if (thread_id < num_vector_elements) {
        const ck_mem::Packed128<const scalar_t> x_vec =
            reinterpret_cast<const ck_mem::Packed128<const scalar_t> *>(x)[thread_id];
        scalar_t output_buffer[num_elements_per_thread];

        // clang-format off
        #pragma unroll
        // clang-format on
        for (uint32 i = 0; i < num_elements_per_thread; i++) {
            output_buffer[i] = x_vec[i] + y;
        }

        ck::memory::store128<scalar_t>(
            output, reinterpret_cast<ck_mem::Packed128<scalar_t> *>(output_buffer)[0], thread_id);
    }

    const uint32 index = num_vector_elements * num_elements_per_thread + thread_id;
    if (index < num_elements) {
        output[index] = x[index] + y;
    }
}

void add_scalar_cuda(const torch::Tensor &x, const float &y, torch::Tensor &output, const uint32 &BLOCK_SIZE) {
    CHECK_CUDA_TENSOR(x);
    CHECK_CUDA_TENSOR(output);

    CHECK_VALID_THREAD_BLOCK(BLOCK_SIZE);

    const uint64 total_elements = x.numel();

    AT_DISPATCH_CUSTOM_FLOAT_TYPES(x.scalar_type(), "add_scalar_cuda_kernel", ([&] {
                                       const uint32 num_elements_per_thread = 16 / sizeof(scalar_t);
                                       const uint32 num_elements_per_block = BLOCK_SIZE * num_elements_per_thread;

                                       std::vector<ck::ChunkedArray<scalar_t>> x_chunks =
                                           ck::chunk_array<scalar_t>(x.data_ptr<scalar_t>(), total_elements);
                                       std::vector<ck::ChunkedArray<scalar_t>> output_chunks =
                                           ck::chunk_array<scalar_t>(output.data_ptr<scalar_t>(), total_elements);

                                       for (int i = 0; i < x_chunks.size(); i++) {
                                           ck::ChunkedArray<scalar_t> x_chunk = x_chunks[i];
                                           ck::ChunkedArray<scalar_t> output_chunk = output_chunks[i];

                                           const uint64 num_elements = x_chunk.num_elements;
                                           const uint32 NUM_BLOCKS =
                                               ck::ceil_divide<uint64>(num_elements, num_elements_per_block);

                                           _add_scalar_cuda_kernel<scalar_t><<<NUM_BLOCKS, BLOCK_SIZE>>>(
                                               x_chunk.array, y, output_chunk.array, num_elements);
                                       }
                                   }));
}
