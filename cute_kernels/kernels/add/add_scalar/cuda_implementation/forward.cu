#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "../../../../include/dtypes/all.h"
#include "../../../../include/launch.h"
#include "../../../../include/math.h"
#include "../../../../include/threads.h"

template <typename scalar_t>
__global__ void _add_scalar_cuda_kernel(const scalar_t *x, const fp32 y, scalar_t *output, const uint64 num_elements) {
    constexpr int num_elements_per_thread = sizeof(fp32_4) / sizeof(scalar_t);
    static_assert(num_elements_per_thread == 4 || num_elements_per_thread == 8);

    using dtype = DType<scalar_t>;
    using T = typename dtype::nv_dtype;
    using T2 = typename dtype::nv_dtype2;

    const uint32 thread_id = get_global_thread_id();
    uint32 end = (thread_id + 1) * num_elements_per_thread - 1;  // inclusive of last element

    if (end < num_elements) {
        const fp32 *x_vec = (fp32 *)&((fp32_4 *)x)[thread_id];
        fp32 output_buffer[4];

        // clang-format off
        #pragma unroll
        // clang-format on
        for (int i = 0; i < 4; i++) {
            if constexpr (std::is_same_v<scalar_t, fp32>) {
                output_buffer[i] = x_vec[i] + y;
            } else {
                fp32_2 _x_upcast = dtype::upcast(dtype::reinterpret_32_bits_as_2x16(x_vec[i]));
                _x_upcast = DType<fp32>::make2(_x_upcast.x + y, _x_upcast.y + y);
                output_buffer[i] = dtype::reinterpret_2x16_as_32_bits(dtype::downcast(_x_upcast));
            }
        }

        ((fp32_4 *)output)[thread_id] = DType<fp32>::make4(output_buffer);
    }

    end = (num_elements / num_elements_per_thread) * num_elements_per_thread + thread_id;
    if (end < num_elements) {
        output[end] = x[end] + y;
    }
}

void add_scalar_cuda(const torch::Tensor &x, const float &y, torch::Tensor &output, const uint32 &BLOCK_SIZE) {
    assert(BLOCK_SIZE % WARP_SIZE == 0);
    const uint64 total_elements = x.numel();

    AT_DISPATCH_CUSTOM_FLOAT_TYPES(
        x.scalar_type(), "add_scalar_cuda_kernel", ([&] {
            const uint32 num_elements_per_thread = 16 / sizeof(scalar_t);
            const uint32 num_elements_per_block = BLOCK_SIZE * num_elements_per_thread;

            std::vector<ChunkedArray<scalar_t>> x_chunks =
                chunk_array<scalar_t>(x.data_ptr<scalar_t>(), total_elements);
            std::vector<ChunkedArray<scalar_t>> output_chunks =
                chunk_array<scalar_t>(output.data_ptr<scalar_t>(), total_elements);

            for (int i = 0; i < x_chunks.size(); i++) {
                ChunkedArray<scalar_t> x_chunk = x_chunks[i];
                ChunkedArray<scalar_t> output_chunk = output_chunks[i];

                const uint64 num_elements = x_chunk.num_elements;
                const uint32 NUM_BLOCKS = ceil_divide<uint64>(num_elements, num_elements_per_block);

                if constexpr (std::is_same_v<scalar_t, fp32>) {
                    _add_scalar_cuda_kernel<scalar_t>
                        <<<NUM_BLOCKS, BLOCK_SIZE>>>(x_chunk.array, y, output_chunk.array, num_elements);
                } else {
                    _add_scalar_cuda_kernel<scalar_t>
                        <<<NUM_BLOCKS, BLOCK_SIZE>>>(x_chunk.array, y, output_chunk.array, num_elements);
                }
            }
        }));
}
