#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "include/cute_kernels.h"

namespace ck = cute_kernels;
namespace ck_mem = ck::memory;

using fp32 = ck::fp32;
using uint32 = ck::uint32;
using uint64 = ck::uint64;
using fp32_2 = ck::fp32_2;

template <typename scalar_t>
__global__ void _add_scalar_cuda_kernel(const scalar_t *x, const fp32 y, scalar_t *output, const uint64 num_elements) {
    using dtype = ck::DType<scalar_t>;
    constexpr uint32 num_elements_per_thread = ck_mem::Packed128<scalar_t>::size;

    const uint32 thread_id = ck::get_global_thread_id();
    const uint32 num_vector_elements = num_elements / num_elements_per_thread;

    if (thread_id < num_vector_elements) {
        // packed array allows loading using vector loads, its just a syntactic sugar
        const ck_mem::Packed128<const scalar_t> x_vec = ck_mem::Packed128Array<const scalar_t>(x)[thread_id];
        ck_mem::Packed128<scalar_t> output_buffer;

        // clang-format off
        #pragma unroll
        // clang-format on
        for (uint32 i = 0; i < 4; i++) {
            if constexpr (std::is_same_v<scalar_t, fp32>) {
                output_buffer[i] = x_vec[i] + y;
            } else {
                using T2 = typename dtype::nv_dtype2;

                const uint32 index = i << 1;

                fp32_2 x2 = dtype::upcast(dtype::make2(x_vec[index], x_vec[index + 1]));
                x2.x = x2.x + y;
                x2.y = x2.y + y;
                T2 output2 = dtype::downcast(x2);

                output_buffer[index] = output2.x;
                output_buffer[index + 1] = output2.y;
            }
        }

        ck_mem::Packed128Array<scalar_t> output_vec = ck_mem::Packed128Array<scalar_t>(output);
        output_vec[thread_id] = output_buffer;
    }

    const uint32 index = num_vector_elements * num_elements_per_thread + thread_id;
    if (index < num_elements) {
        output[index] = x[index] + y;
    }
}

void add_scalar_cuda(const torch::Tensor &x, const fp32 &y, torch::Tensor &output, const uint32 &BLOCK_SIZE) {
    CHECK_CUDA_TENSOR(x);
    CHECK_CUDA_TENSOR(output);

    CHECK_VALID_THREAD_BLOCK(BLOCK_SIZE);

    const uint64 total_elements = x.numel();

    AT_DISPATCH_CUSTOM_FLOAT_TYPES(x.scalar_type(), "add_scalar_cuda_kernel", ([&] {
                                       const uint32 num_elements_per_thread = 16 / sizeof(scalar_t);
                                       const uint32 num_elements_per_block = BLOCK_SIZE * num_elements_per_thread;

                                       std::vector<ck::ChunkedArray<scalar_t>> x_chunks =
                                           ck::chunk_array<scalar_t>(x.data_ptr<scalar_t>(), total_elements);
                                       std::vector<ck::ChunkedArray<scalar_t>> output_chunks =
                                           ck::chunk_array<scalar_t>(output.data_ptr<scalar_t>(), total_elements);

                                       for (int i = 0; i < x_chunks.size(); i++) {
                                           ck::ChunkedArray<scalar_t> x_chunk = x_chunks[i];
                                           ck::ChunkedArray<scalar_t> output_chunk = output_chunks[i];

                                           const uint64 num_elements = x_chunk.num_elements;
                                           const uint32 NUM_BLOCKS =
                                               ck::ceil_divide<uint64>(num_elements, num_elements_per_block);

                                           _add_scalar_cuda_kernel<scalar_t><<<NUM_BLOCKS, BLOCK_SIZE>>>(
                                               x_chunk.array, y, output_chunk.array, num_elements);
                                       }
                                   }));
}
