#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "../../../include/dtypes/all.h"
#include "../../../include/launch.h"
#include "../../../include/math.h"
#include "../../../include/threads.h"

#define MAX_ALLOWED_C 16384

namespace cg = cooperative_groups;

inline __device__ void _looped_atomic_add(uint32 *output_shared,
                                          uint32 *destination_output_shared,
                                          const uint32 &num_loops_C,
                                          const uint32 &C,
                                          const uint32 &local_thread_id) {
    for (int i = 0; i < num_loops_C; i++) {
        const int index = i * blockDim.x + local_thread_id;
        if (index < C) {
            atomicAdd(&destination_output_shared[index], output_shared[index]);
        }
    }
}

inline __device__ void _initialize_global_output(uint32 *output, const uint32 &C, const uint32 &global_thread_id) {
    for (uint32 i = global_thread_id; i < C >> 2; i += gridDim.x * blockDim.x) {
        ((uint32_4 *)output)[i] = DType<uint32>::make4(0, 0, 0, 0);
    }

    if (global_thread_id < C - ((C >> 2) << 2)) {
        output[global_thread_id] = 0;
    }
}

template <typename scalar_t>
__global__ void _continuous_count_cuda_kernel(
    const scalar_t *x, uint32 *output, const uint64 num_elements, const uint32 C, const bool initialize_output) {
    const uint32 local_thread_id = get_local_thread_id();
    const uint32 global_thread_id = get_global_thread_id();
    const uint32 num_loops_C = ceil_divide<uint32>(C, blockDim.x);

    extern __shared__ uint32 output_shared[];

    for (uint32 i = 0; i < num_loops_C; i++) {
        const uint32 index = i * blockDim.x + local_thread_id;
        if (index < C) {
            output_shared[index] = 0;
        }
    }

    if (initialize_output) {
        _initialize_global_output(output, C, global_thread_id);
        cg::this_grid().sync();
    }

    for (uint32 i = global_thread_id; i < num_elements; i += gridDim.x * blockDim.x) {
        atomicAdd(&output_shared[x[i]], 1);
    }

    cg::cluster_group cluster = cg::this_cluster();
    const bool is_first_cluster_block = cluster.block_rank() == 0;

    __syncthreads();

    if (!is_first_cluster_block) {
        _looped_atomic_add(output_shared, cluster.map_shared_rank(output_shared, 0), num_loops_C, C, local_thread_id);
    }

    cluster.sync();

    // write the output to the global memory
    if (is_first_cluster_block) {
        _looped_atomic_add(output_shared, output, num_loops_C, C, local_thread_id);
    }
}

void continuous_count_cuda(const torch::Tensor &x,
                           torch::Tensor &output,
                           const uint32 &sm_count,
                           const uint32 &thread_block_cluster_size,
                           const uint32 &C,
                           const uint32 &BLOCK_SIZE) {
    assert(BLOCK_SIZE % WARP_SIZE == 0);
    assert(C <= MAX_ALLOWED_C);

    const uint64 total_elements = x.numel();
    const int max_num_blocks = get_max_thread_blocks(sm_count, thread_block_cluster_size);

    std::vector<ChunkedArray<uint32>> output_chunks = chunk_array<uint32>(output.data_ptr<uint32>(), total_elements);

    AT_DISPATCH_CUSTOM_INT_TYPES(x.scalar_type(), "continuous_count_cuda_kernel", ([&] {
                                     hipFuncSetAttribute(reinterpret_cast<const void*>(_continuous_count_cuda_kernel<scalar_t>),
                                                          hipFuncAttributeMaxDynamicSharedMemorySize,
                                                          MAX_ALLOWED_C * sizeof(uint32));

                                     std::vector<ChunkedArray<scalar_t>> x_chunks =
                                         chunk_array<scalar_t>(x.data_ptr<scalar_t>(), total_elements);

                                     for (int i = 0; i < x_chunks.size(); i++) {
                                         ChunkedArray<scalar_t> x_chunk = x_chunks[i];
                                         ChunkedArray<uint32> output_chunk = output_chunks[i];

                                         const uint64 num_elements = x_chunk.num_elements;

                                         auto [NUM_BLOCKS, cluster_size] = get_num_blocks(
                                             num_elements, BLOCK_SIZE, max_num_blocks, thread_block_cluster_size);

                                         // dynamically sized clusters need this stupid way of launching the kernel
                                         hipLaunchConfig_t launch_config = {0};
                                         launch_config.blockDim = BLOCK_SIZE;
                                         launch_config.gridDim = NUM_BLOCKS;
                                         launch_config.dynamicSmemBytes = C * sizeof(uint32);

                                         hipLaunchAttribute attributes[2];

                                         attributes[0].id = cudaLaunchAttributeClusterDimension;
                                         attributes[0].val.clusterDim.x = cluster_size;
                                         attributes[0].val.clusterDim.y = 1;
                                         attributes[0].val.clusterDim.z = 1;

                                         attributes[1].id = hipLaunchAttributeCooperative;
                                         attributes[1].val.cooperative = 1;

                                         launch_config.attrs = attributes;
                                         launch_config.numAttrs = 2;

                                         cudaLaunchKernelEx(&launch_config,
                                                            _continuous_count_cuda_kernel<scalar_t>,
                                                            x_chunk.array,
                                                            output_chunk.array,
                                                            num_elements,
                                                            C,
                                                            i == 0);
                                     }
                                 }));
}
