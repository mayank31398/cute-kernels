#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "include/cute_kernels.h"

namespace ck = cute_kernels;
namespace ck_mem = ck::memory;

using fp32 = ck::fp32;
using uint32 = ck::uint32;
using uint64 = ck::uint64;

template <typename scalar_t, bool has_trailing_elements>
__global__ void _add_scalar_cuda_kernel(const scalar_t *x, const fp32 y, scalar_t *output, const uint64 num_elements) {
    constexpr uint32 num_elements_per_thread = ck_mem::get_num_elements_for_vector_load_stores<scalar_t>();

    const uint32 thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32 num_vector_elements = num_elements / num_elements_per_thread;

    if (thread_id < num_vector_elements) {
        const scalar_t *x_vec = ck_mem::load_128_bits<const scalar_t>(x, thread_id);
        scalar_t output_buffer[num_elements_per_thread];

        for (uint32 i = 0; i < num_elements_per_thread; i++) {
            output_buffer[i] = x_vec[i] + y;
        }

        ck_mem::store_128_bits<scalar_t>(output_buffer, output, thread_id);
    }

    if (has_trailing_elements) {
        const uint32 warp_id = thread_id >> LOG_WARP_SIZE;
        const uint32 num_warps = (gridDim.x * blockDim.x) >> LOG_WARP_SIZE;
        const bool is_last_warp = warp_id == num_warps - 1;

        if (is_last_warp) {
            const uint32 index = num_vector_elements * num_elements_per_thread + (threadIdx.x % WARP_SIZE);
            if (index < num_elements) {
                output[index] = x[index] + y;
            }
        }
    }
}

void add_scalar_cuda(const torch::Tensor &x, const fp32 &y, torch::Tensor &output, const uint32 &BLOCK_SIZE) {
    CHECK_CUDA_TENSOR(x);
    CHECK_CUDA_TENSOR(output);

    CHECK_VALID_THREAD_BLOCK(BLOCK_SIZE);

    const uint64 total_elements = x.numel();

    AT_DISPATCH_CUSTOM_FLOAT_TYPES(
        x.scalar_type(), "add_scalar_cuda_kernel", ([&] {
            const uint32 num_elements_per_thread = 16 / sizeof(scalar_t);
            const uint32 num_elements_per_block = num_elements_per_thread * BLOCK_SIZE;

            std::vector<ck::ChunkedArray<scalar_t>> x_chunks =
                ck::chunk_array<scalar_t>(x.data_ptr<scalar_t>(), total_elements);
            std::vector<ck::ChunkedArray<scalar_t>> output_chunks =
                ck::chunk_array<scalar_t>(output.data_ptr<scalar_t>(), total_elements);

            for (int i = 0; i < x_chunks.size(); i++) {
                ck::ChunkedArray<scalar_t> x_chunk = x_chunks[i];
                ck::ChunkedArray<scalar_t> output_chunk = output_chunks[i];

                const uint64 num_elements = x_chunk.num_elements;
                constexpr bool has_trailing_elements = ck::convert_bool_to_static_bool(
                    (i == x_chunks.size() - 1) && (num_elements % num_elements_per_thread != 0));

                uint32 NUM_BLOCKS;
                if (has_trailing_elements) {
                    const uint32 num_elements_per_warp = num_elements_per_thread << LOG_WARP_SIZE;
                    const uint32 num_warps_per_block = BLOCK_SIZE >> LOG_WARP_SIZE;
                    // 1 extra warp to avoid thread divergence
                    const uint32 NUM_WARPS = ck::ceil_divide<uint64>(num_elements, num_elements_per_warp) + 1;
                    NUM_BLOCKS = ck::ceil_divide<uint64>(NUM_WARPS, num_warps_per_block);
                } else {
                    NUM_BLOCKS = ck::ceil_divide<uint64>(num_elements, num_elements_per_block);
                }

                _add_scalar_cuda_kernel<scalar_t, has_trailing_elements>
                    <<<NUM_BLOCKS, BLOCK_SIZE>>>(x_chunk.array, y, output_chunk.array, num_elements);
            }
        }));
}
