#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "include/cute_kernels.h"

namespace ck = cute_kernels;
namespace ck_mem = ck::memory;

using fp32 = ck::fp32;
using uint32 = ck::uint32;
using uint64 = ck::uint64;

template <typename scalar_t, bool has_trailing_elements>
__global__ void add_scalar_cuda_kernel(const scalar_t *x, const fp32 y, scalar_t *output, const uint64 N) {
    constexpr uint32 num_elements_per_thread = ck_mem::get_num_elements_for_vector_load_stores<scalar_t>();

    const uint32 thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32 N_vector = N / num_elements_per_thread;

    if (thread_id < N_vector) {
        const scalar_t *x_vec = ck_mem::vectorized_load<const scalar_t>(x, thread_id);
        scalar_t output_buffer[num_elements_per_thread];

        for (uint32 i = 0; i < num_elements_per_thread; i++) {
            output_buffer[i] = x_vec[i] + y;
        }

        ck_mem::vectorized_store<scalar_t>(output_buffer, output, thread_id);
    }

    if (has_trailing_elements) {
        const uint32 warp_id = thread_id >> LOG_WARP_SIZE;
        const uint32 num_warps = (gridDim.x * blockDim.x) >> LOG_WARP_SIZE;
        const bool is_last_warp = warp_id == num_warps - 1;

        if (is_last_warp) {
            const uint32 index = N_vector * num_elements_per_thread + (threadIdx.x % WARP_SIZE);
            if (index < N) {
                output[index] = x[index] + y;
            }
        }
    }
}

void add_scalar_cuda(const torch::Tensor &x, const fp32 &y, torch::Tensor &output, const uint32 &BLOCK_SIZE) {
    CHECK_CUDA_TENSOR(x);
    CHECK_CUDA_TENSOR(output);

    CHECK_VALID_THREAD_BLOCK(BLOCK_SIZE);

    const uint64 total_elements = x.numel();

    AT_DISPATCH_CUSTOM_FLOAT_TYPES(
        x.scalar_type(), "add_scalar_cuda_kernel", ([&] {
            std::vector<ck::ChunkedArray<scalar_t>> x_chunks =
                ck::chunk_array<scalar_t>(x.data_ptr<scalar_t>(), total_elements);
            std::vector<ck::ChunkedArray<scalar_t>> output_chunks =
                ck::chunk_array<scalar_t>(output.data_ptr<scalar_t>(), total_elements);

            for (int i = 0; i < x_chunks.size(); i++) {
                ck::ChunkedArray<scalar_t> x_chunk = x_chunks[i];
                ck::ChunkedArray<scalar_t> output_chunk = output_chunks[i];

                const uint64 num_elements = x_chunk.num_elements;

                constexpr uint32 bits = 32;
                const uint32 num_elements_per_thread =
                    ck_mem::get_num_elements_for_vector_load_stores<scalar_t, bits>();
                const uint32 num_elements_per_block = BLOCK_SIZE * num_elements_per_thread;

                const bool has_trailing_elements =
                    (i == x_chunks.size() - 1) && (num_elements % num_elements_per_thread != 0);

                if (has_trailing_elements) {
                    const uint32 num_elements_per_warp = num_elements_per_thread << LOG_WARP_SIZE;
                    const uint32 num_warps_per_block = BLOCK_SIZE >> LOG_WARP_SIZE;
                    // 1 extra warp to avoid thread divergence
                    const uint32 NUM_WARPS = ck::ceil_divide<uint64>(num_elements, num_elements_per_warp) + 1;
                    const uint32 NUM_BLOCKS = ck::ceil_divide<uint64>(NUM_WARPS, num_warps_per_block);

                    add_scalar_cuda_kernel<scalar_t, true, bits>
                        <<<NUM_BLOCKS, BLOCK_SIZE>>>(x_chunk.array, y, output_chunk.array, num_elements);
                } else {
                    const uint32 NUM_BLOCKS = ck::ceil_divide<uint64>(num_elements, num_elements_per_block);

                    add_scalar_cuda_kernel<scalar_t, false, bits>
                        <<<NUM_BLOCKS, BLOCK_SIZE>>>(x_chunk.array, y, output_chunk.array, num_elements);
                }
            }
        }));
}
