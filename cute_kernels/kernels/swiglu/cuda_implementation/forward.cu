#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "include/cute_kernels.h"

namespace ck = cute_kernels;
namespace ck_mem = ck::memory;

using fp32 = ck::fp32;
using uint32 = ck::uint32;
using uint64 = ck::uint64;

template <typename scalar_t>
inline __device__ scalar_t _swiglu_forward(const scalar_t &gate, const scalar_t &up) {
    using dtype = ck::DType<scalar_t>;

    fp32 _up = dtype::upcast(up);
    fp32 _gate = dtype::upcast(gate);
    fp32 _sigmoid = ck::sigmoid<fp32, fp32>(_gate);

    _sigmoid *= _gate * _up;

    return dtype::downcast(_sigmoid);
}

template <typename scalar_t>
__global__ void _swiglu_forward_cuda_kernel(const scalar_t *gate,
                                            const scalar_t *up,
                                            scalar_t *output,
                                            const uint64 num_elements) {
    constexpr uint32 num_elements_per_thread = ck_mem::get_num_elements_for_vector_load_stores<scalar_t>();

    const uint32 thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32 num_vector_elements = num_elements / num_elements_per_thread;

    if (thread_id < num_vector_elements) {
        const scalar_t *gate_vec = ck_mem::vectorized_load<const scalar_t>(gate, thread_id);
        const scalar_t *up_vec = ck_mem::vectorized_load<const scalar_t>(up, thread_id);
        scalar_t output_buffer[num_elements_per_thread];

        for (uint32 i = 0; i < num_elements_per_thread; i++) {
            output_buffer[i] = _swiglu_forward<scalar_t>(gate_vec[i], up_vec[i]);
        }

        ck_mem::vectorized_store<scalar_t>(output_buffer, output, thread_id);
    }

    const uint32 index = num_vector_elements * num_elements_per_thread + thread_id;
    if (index < num_elements) {
        output[index] = _swiglu_forward<scalar_t>(gate[index], up[index]);
    }
}

void swiglu_forward_cuda(const torch::Tensor &gate,
                         const torch::Tensor &up,
                         torch::Tensor &output,
                         const uint32 &BLOCK_SIZE) {
    CHECK_CUDA_TENSOR(gate);
    CHECK_CUDA_TENSOR(up);
    CHECK_CUDA_TENSOR(output);

    CHECK_VALID_THREAD_BLOCK(BLOCK_SIZE);

    const uint64 total_elements = gate.numel();

    AT_DISPATCH_CUSTOM_FLOAT_TYPES(gate.scalar_type(), "swiglu_forward_cuda_kernel", ([&] {
                                       const uint32 num_elements_per_thread = 16 / sizeof(scalar_t);
                                       const uint32 num_elements_per_block = BLOCK_SIZE * num_elements_per_thread;

                                       std::vector<ck::ChunkedArray<scalar_t>> gate_chunks =
                                           ck::chunk_array<scalar_t>(gate.data_ptr<scalar_t>(), total_elements);
                                       std::vector<ck::ChunkedArray<scalar_t>> up_chunks =
                                           ck::chunk_array<scalar_t>(up.data_ptr<scalar_t>(), total_elements);
                                       std::vector<ck::ChunkedArray<scalar_t>> output_chunks =
                                           ck::chunk_array<scalar_t>(output.data_ptr<scalar_t>(), total_elements);

                                       for (int i = 0; i < gate_chunks.size(); i++) {
                                           ck::ChunkedArray<scalar_t> gate_chunk = gate_chunks[i];
                                           ck::ChunkedArray<scalar_t> up_chunk = up_chunks[i];
                                           ck::ChunkedArray<scalar_t> output_chunk = output_chunks[i];

                                           const uint64 num_elements = gate_chunk.num_elements;
                                           const uint32 NUM_BLOCKS =
                                               ck::ceil_divide<uint64>(num_elements, num_elements_per_block);

                                           _swiglu_forward_cuda_kernel<scalar_t><<<NUM_BLOCKS, BLOCK_SIZE>>>(
                                               gate_chunk.array, up_chunk.array, output_chunk.array, num_elements);
                                       }
                                   }));
}
