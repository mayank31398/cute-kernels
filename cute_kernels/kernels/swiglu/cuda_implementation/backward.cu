#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "include/cute_kernels.h"

namespace ck = cute_kernels;
namespace ck_mem = ck::memory;

using fp32 = ck::fp32;
using uint32 = ck::uint32;
using uint64 = ck::uint64;

template <typename scalar_t>
inline __device__ void _swiglu_backward(const scalar_t &gate,
                                        const scalar_t &up,
                                        const scalar_t &output_grad,
                                        scalar_t *gate_grad_buffer,
                                        scalar_t *up_grad_buffer,
                                        const uint32 &index) {
    using dtype = ck::DType<scalar_t>;

    fp32 _gate = dtype::upcast(gate);
    fp32 _up = dtype::upcast(up);
    fp32 _output_grad = dtype::upcast(output_grad);

    fp32 _gate_sigmoid = ck::sigmoid<fp32, fp32>(_gate);
    fp32 _gate_silu = _gate * _gate_sigmoid;

    fp32 _gate_grad = _output_grad * _up * (_gate_sigmoid + _gate_silu * (1 - _gate_sigmoid));
    fp32 _up_grad = _output_grad * _gate_silu;

    scalar_t gate_grad = dtype::downcast(_gate_grad);
    scalar_t up_grad = dtype::downcast(_up_grad);

    gate_grad_buffer[index] = gate_grad;
    up_grad_buffer[index] = up_grad;
}

template <typename scalar_t, bool has_trailing_elements>
__global__ void _swiglu_backward_cuda_kernel(const scalar_t *gate,
                                             const scalar_t *up,
                                             const scalar_t *output_grad,
                                             scalar_t *gate_grad,
                                             scalar_t *up_grad,
                                             const uint64 num_elements) {
    constexpr uint32 num_elements_per_thread = ck_mem::get_num_elements_for_vector_load_stores<scalar_t>();

    const uint32 thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32 num_vector_elements = num_elements / num_elements_per_thread;

    if (thread_id < num_vector_elements) {
        const scalar_t *gate_vec = ck_mem::load_128_bits<const scalar_t>(gate, thread_id);
        const scalar_t *up_vec = ck_mem::load_128_bits<const scalar_t>(up, thread_id);
        const scalar_t *output_grad_vec = ck_mem::load_128_bits<const scalar_t>(output_grad, thread_id);

        scalar_t gate_grad_buffer[num_elements_per_thread];
        scalar_t up_grad_buffer[num_elements_per_thread];

        for (uint32 i = 0; i < num_elements_per_thread; i++) {
            _swiglu_backward<scalar_t>(
                gate_vec[i], up_vec[i], output_grad_vec[i], gate_grad_buffer, up_grad_buffer, i);
        }

        ck_mem::store_128_bits<scalar_t>(gate_grad_buffer, gate_grad, thread_id);
        ck_mem::store_128_bits<scalar_t>(up_grad_buffer, up_grad, thread_id);
    }

    if (has_trailing_elements) {
        const uint32 warp_id = thread_id >> LOG_WARP_SIZE;
        const uint32 num_warps = (gridDim.x * blockDim.x) >> LOG_WARP_SIZE;
        const bool is_last_warp = warp_id == num_warps - 1;

        if (is_last_warp) {
            const uint32 index = num_vector_elements * num_elements_per_thread + (threadIdx.x % WARP_SIZE);
            if (index < num_elements) {
                _swiglu_backward<scalar_t>(gate[index], up[index], output_grad[index], gate_grad, up_grad, index);
            }
        }
    }
}

void swiglu_backward_cuda(const torch::Tensor &gate,
                          const torch::Tensor &up,
                          const torch::Tensor &output_grad,
                          torch::Tensor &gate_grad,
                          torch::Tensor &up_grad,
                          const uint32 &BLOCK_SIZE) {
    CHECK_CUDA_TENSOR(gate);
    CHECK_CUDA_TENSOR(up);
    CHECK_CUDA_TENSOR(output_grad);
    CHECK_CUDA_TENSOR(gate_grad);
    CHECK_CUDA_TENSOR(up_grad);

    CHECK_VALID_THREAD_BLOCK(BLOCK_SIZE);

    const uint64 total_elements = gate.numel();

    AT_DISPATCH_CUSTOM_FLOAT_TYPES(
        gate.scalar_type(), "swiglu_backward_cuda_kernel", ([&] {
            const uint32 num_elements_per_thread = 16 / sizeof(scalar_t);
            const uint32 num_elements_per_block = num_elements_per_thread * BLOCK_SIZE;

            std::vector<ck::ChunkedArray<scalar_t>> gate_chunks =
                ck::chunk_array<scalar_t>(gate.data_ptr<scalar_t>(), total_elements);
            std::vector<ck::ChunkedArray<scalar_t>> up_chunks =
                ck::chunk_array<scalar_t>(up.data_ptr<scalar_t>(), total_elements);
            std::vector<ck::ChunkedArray<scalar_t>> output_grad_chunks =
                ck::chunk_array<scalar_t>(output_grad.data_ptr<scalar_t>(), total_elements);
            std::vector<ck::ChunkedArray<scalar_t>> gate_grad_chunks =
                ck::chunk_array<scalar_t>(gate_grad.data_ptr<scalar_t>(), total_elements);
            std::vector<ck::ChunkedArray<scalar_t>> up_grad_chunks =
                ck::chunk_array<scalar_t>(up_grad.data_ptr<scalar_t>(), total_elements);

            for (int i = 0; i < gate_chunks.size(); i++) {
                ck::ChunkedArray<scalar_t> gate_chunk = gate_chunks[i];
                ck::ChunkedArray<scalar_t> up_chunk = up_chunks[i];
                ck::ChunkedArray<scalar_t> output_grad_chunk = output_grad_chunks[i];
                ck::ChunkedArray<scalar_t> gate_grad_chunk = gate_grad_chunks[i];
                ck::ChunkedArray<scalar_t> up_grad_chunk = up_grad_chunks[i];

                const uint64 num_elements = gate_chunk.num_elements;
                const bool has_trailing_elements =
                    (i == x_chunks.size() - 1) && (num_elements % num_elements_per_thread != 0);

                if (has_trailing_elements) {
                    const uint32 num_elements_per_warp = num_elements_per_thread << LOG_WARP_SIZE;
                    const uint32 num_warps_per_block = BLOCK_SIZE >> LOG_WARP_SIZE;
                    // 1 extra warp to avoid thread divergence
                    const uint32 NUM_WARPS = ck::ceil_divide<uint64>(num_elements, num_elements_per_warp) + 1;
                    const uint32 NUM_BLOCKS = ck::ceil_divide<uint64>(NUM_WARPS, num_warps_per_block);

                    _swiglu_backward_cuda_kernel<scalar_t, true><<<NUM_BLOCKS, BLOCK_SIZE>>>(gate_chunk.array,
                                                                                             up_chunk.array,
                                                                                             output_grad_chunk.array,
                                                                                             gate_grad_chunk.array,
                                                                                             up_grad_chunk.array,
                                                                                             num_elements);
                } else {
                    const uint32 NUM_BLOCKS = ck::ceil_divide<uint64>(num_elements, num_elements_per_block);

                    _swiglu_backward_cuda_kernel<scalar_t, false><<<NUM_BLOCKS, BLOCK_SIZE>>>(gate_chunk.array,
                                                                                              up_chunk.array,
                                                                                              output_grad_chunk.array,
                                                                                              gate_grad_chunk.array,
                                                                                              up_grad_chunk.array,
                                                                                              num_elements);
                }
            }
        }));
}
