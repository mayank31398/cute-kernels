#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "include/activations.h"
#include "include/dtypes.h"
#include "include/launch.h"
#include "include/math.h"
#include "include/threads.h"

namespace ck = cute_kernels;

using uint32 = ck::uint32;
using uint64 = ck::uint64;

using fp32 = ck::fp32;
using fp32_2 = ck::fp32_2;
using fp32_4 = ck::fp32_4;

template <typename scalar_t>
__global__ void _swiglu_backward_cuda_kernel(const scalar_t *gate,
                                             const scalar_t *up,
                                             const scalar_t *output_grad,
                                             scalar_t *gate_grad,
                                             scalar_t *up_grad,
                                             const uint32 num_elements) {
    constexpr int num_elements_per_thread = 16 / sizeof(scalar_t);
    static_assert(num_elements_per_thread == 4 || num_elements_per_thread == 8);

    using dtype = ck::DType<scalar_t>;

    const uint32 thread_id = ck::get_global_thread_id();
    const uint32 num_elements4 = num_elements / num_elements_per_thread;

    if (thread_id < num_elements4) {
        const fp32 *gate_vec = (fp32 *)&((fp32_4 *)gate)[thread_id];
        const fp32 *up_vec = (fp32 *)&((fp32_4 *)up)[thread_id];
        const fp32 *output_grad_vec = (fp32 *)&((fp32_4 *)output_grad)[thread_id];

        fp32 gate_grad_buffer[4];
        fp32 up_grad_buffer[4];

        // clang-format off
        #pragma unroll
        // clang-format on
        for (int i = 0; i < 4; i++) {
            if constexpr (std::is_same_v<scalar_t, fp32>) {
                fp32 _gate_sigmoid = ck::sigmoid<fp32, fp32>(gate_vec[i]);
                fp32 _gate_silu = gate_vec[i] * _gate_sigmoid;

                gate_grad_buffer[i] =
                    output_grad_vec[i] * up_vec[i] * (_gate_sigmoid + _gate_silu * (1 - _gate_sigmoid));
                up_grad_buffer[i] = output_grad_vec[i] * _gate_silu;
            } else {
                fp32_2 _gate_upcast = dtype::upcast(dtype::reinterpret_32_bits_as_2x16(gate_vec[i]));
                fp32_2 _up_upcast = dtype::upcast(dtype::reinterpret_32_bits_as_2x16(up_vec[i]));
                fp32_2 _output_grad_upcast = dtype::upcast(dtype::reinterpret_32_bits_as_2x16(output_grad_vec[i]));

                fp32 _gate_sigmoid_x = ck::sigmoid<fp32, fp32>(_gate_upcast.x);
                fp32 _gate_sigmoid_y = ck::sigmoid<fp32, fp32>(_gate_upcast.y);

                fp32 _gate_silu_x = _gate_upcast.x * _gate_sigmoid_x;
                fp32 _gate_silu_y = _gate_upcast.y * _gate_sigmoid_y;

                _gate_upcast = ck::DType<fp32>::make2(
                    _output_grad_upcast.x * _up_upcast.x * (_gate_sigmoid_x + _gate_silu_x * (1 - _gate_sigmoid_x)),
                    _output_grad_upcast.y * _up_upcast.y * (_gate_sigmoid_y + _gate_silu_y * (1 - _gate_sigmoid_y)));

                _up_upcast =
                    ck::DType<fp32>::make2(_output_grad_upcast.x * _gate_silu_x, _output_grad_upcast.y * _gate_silu_y);

                gate_grad_buffer[i] = dtype::reinterpret_2x16_as_32_bits(dtype::downcast(_gate_upcast));
                up_grad_buffer[i] = dtype::reinterpret_2x16_as_32_bits(dtype::downcast(_up_upcast));
            }
        }

        ((fp32_4 *)gate_grad)[thread_id] = ck::DType<fp32>::make4(gate_grad_buffer);
        ((fp32_4 *)up_grad)[thread_id] = ck::DType<fp32>::make4(up_grad_buffer);
    }

    const uint32 index = num_elements4 * num_elements_per_thread + thread_id;
    if (index < num_elements) {
        fp32 _gate_upcast = dtype::upcast(gate[index]);

        fp32 _gate_sigmoid = ck::sigmoid<fp32, fp32>(_gate_upcast);
        fp32 _gate_silu = _gate_upcast * _gate_sigmoid;

        gate_grad[index] =
            dtype::downcast(output_grad[index] * up[index] * (_gate_sigmoid + _gate_silu * (1 - _gate_sigmoid)));
        up_grad[index] = dtype::downcast(output_grad[index] * _gate_silu);
    }
}

void swiglu_backward_cuda(const torch::Tensor &gate,
                          const torch::Tensor &up,
                          const torch::Tensor &output_grad,
                          torch::Tensor &gate_grad,
                          torch::Tensor &up_grad,
                          const uint32 &BLOCK_SIZE) {
    TORCH_CHECK(gate.is_cuda());
    TORCH_CHECK(up.is_cuda());
    TORCH_CHECK(output_grad.is_cuda());
    TORCH_CHECK(gate_grad.is_cuda());
    TORCH_CHECK(up_grad.is_cuda());

    TORCH_CHECK(BLOCK_SIZE % WARP_SIZE == 0);

    const uint64 total_elements = gate.numel();

    AT_DISPATCH_CUSTOM_FLOAT_TYPES(
        gate.scalar_type(), "swiglu_backward_cuda_kernel", ([&] {
            const uint32 num_elements_per_thread = 16 / sizeof(scalar_t);
            const uint32 num_elements_per_block = BLOCK_SIZE * num_elements_per_thread;

            std::vector<ck::ChunkedArray<scalar_t>> gate_chunks =
                ck::chunk_array<scalar_t>(gate.data_ptr<scalar_t>(), total_elements);
            std::vector<ck::ChunkedArray<scalar_t>> up_chunks =
                ck::chunk_array<scalar_t>(up.data_ptr<scalar_t>(), total_elements);
            std::vector<ck::ChunkedArray<scalar_t>> output_grad_chunks =
                ck::chunk_array<scalar_t>(output_grad.data_ptr<scalar_t>(), total_elements);
            std::vector<ck::ChunkedArray<scalar_t>> gate_grad_chunks =
                ck::chunk_array<scalar_t>(gate_grad.data_ptr<scalar_t>(), total_elements);
            std::vector<ck::ChunkedArray<scalar_t>> up_grad_chunks =
                ck::chunk_array<scalar_t>(up_grad.data_ptr<scalar_t>(), total_elements);

            for (int i = 0; i < gate_chunks.size(); i++) {
                ck::ChunkedArray<scalar_t> gate_chunk = gate_chunks[i];
                ck::ChunkedArray<scalar_t> up_chunk = up_chunks[i];
                ck::ChunkedArray<scalar_t> output_grad_chunk = output_grad_chunks[i];
                ck::ChunkedArray<scalar_t> gate_grad_chunk = gate_grad_chunks[i];
                ck::ChunkedArray<scalar_t> up_grad_chunk = up_grad_chunks[i];

                const uint32 num_elements = gate_chunk.num_elements;
                const uint32 NUM_BLOCKS = ck::ceil_divide<uint64>(num_elements, num_elements_per_block);

                _swiglu_backward_cuda_kernel<scalar_t><<<NUM_BLOCKS, BLOCK_SIZE>>>(gate_chunk.array,
                                                                                   up_chunk.array,
                                                                                   output_grad_chunk.array,
                                                                                   gate_grad_chunk.array,
                                                                                   up_grad_chunk.array,
                                                                                   num_elements);
            }
        }));
}
