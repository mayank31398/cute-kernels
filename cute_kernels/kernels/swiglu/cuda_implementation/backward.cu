#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "include/cute_kernels.h"

namespace ck = cute_kernels;
namespace ck_mem = ck::memory;

using fp32 = ck::fp32;
using uint32 = ck::uint32;
using uint64 = ck::uint64;

template <typename scalar_t>
inline __device__ void _swiglu_backward(const scalar_t &gate,
                                        const scalar_t &up,
                                        const scalar_t &output_grad,
                                        scalar_t *gate_grad_buffer,
                                        scalar_t *up_grad_buffer,
                                        const uint32 &index) {
    using dtype = ck::DType<scalar_t>;

    fp32 _gate = dtype::upcast(gate);
    fp32 _up = dtype::upcast(up);
    fp32 _output_grad = dtype::upcast(output_grad);

    fp32 _gate_sigmoid = ck::sigmoid<fp32, fp32>(_gate);
    fp32 _gate_silu = _gate * _gate_sigmoid;

    fp32 _gate_grad = _output_grad * _up * (_gate_sigmoid + _gate_silu * (1 - _gate_sigmoid));
    fp32 _up_grad = _output_grad * _gate_silu;

    scalar_t gate_grad = dtype::downcast(_gate_grad);
    scalar_t up_grad = dtype::downcast(_up_grad);

    gate_grad_buffer[index] = gate_grad;
    up_grad_buffer[index] = up_grad;
}

template <typename scalar_t>
__global__ void _swiglu_backward_cuda_kernel(const scalar_t *gate,
                                             const scalar_t *up,
                                             const scalar_t *output_grad,
                                             scalar_t *gate_grad,
                                             scalar_t *up_grad,
                                             const uint64 num_elements) {
    constexpr uint32 num_elements_per_thread = ck_mem::Packed128<scalar_t>::size;

    const uint32 thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32 num_vector_elements = num_elements / num_elements_per_thread;

    if (thread_id < num_vector_elements) {
        const scalar_t *gate_vec = ck_mem::load_128_bits<const scalar_t>(gate, thread_id);
        const scalar_t *up_vec = ck_mem::load_128_bits<const scalar_t>(up, thread_id);
        const scalar_t *output_grad_vec = ck_mem::load_128_bits<const scalar_t>(output_grad, thread_id);

        scalar_t gate_grad_buffer[num_elements_per_thread];
        scalar_t up_grad_buffer[num_elements_per_thread];

        for (uint32 i = 0; i < num_elements_per_thread; i++) {
            _swiglu_backward<scalar_t>(
                gate_vec[i], up_vec[i], output_grad_vec[i], gate_grad_buffer, up_grad_buffer, i);
        }

        ck_mem::Packed128Array<scalar_t> gate_grad_vec = ck_mem::Packed128Array<scalar_t>(gate_grad);
        gate_grad_vec[thread_id] = gate_grad_buffer;

        ck_mem::Packed128Array<scalar_t> up_grad_vec = ck_mem::Packed128Array<scalar_t>(up_grad);
        up_grad_vec[thread_id] = up_grad_buffer;
    }

    const uint32 index = num_vector_elements * num_elements_per_thread + thread_id;
    if (index < num_elements) {
        _swiglu_backward<scalar_t>(gate[index], up[index], output_grad[index], gate_grad, up_grad, index);
    }
}

void swiglu_backward_cuda(const torch::Tensor &gate,
                          const torch::Tensor &up,
                          const torch::Tensor &output_grad,
                          torch::Tensor &gate_grad,
                          torch::Tensor &up_grad,
                          const uint32 &BLOCK_SIZE) {
    CHECK_CUDA_TENSOR(gate);
    CHECK_CUDA_TENSOR(up);
    CHECK_CUDA_TENSOR(output_grad);
    CHECK_CUDA_TENSOR(gate_grad);
    CHECK_CUDA_TENSOR(up_grad);

    CHECK_VALID_THREAD_BLOCK(BLOCK_SIZE);

    const uint64 total_elements = gate.numel();

    AT_DISPATCH_CUSTOM_FLOAT_TYPES(
        gate.scalar_type(), "swiglu_backward_cuda_kernel", ([&] {
            const uint32 num_elements_per_thread = 16 / sizeof(scalar_t);
            const uint32 num_elements_per_block = BLOCK_SIZE * num_elements_per_thread;

            std::vector<ck::ChunkedArray<scalar_t>> gate_chunks =
                ck::chunk_array<scalar_t>(gate.data_ptr<scalar_t>(), total_elements);
            std::vector<ck::ChunkedArray<scalar_t>> up_chunks =
                ck::chunk_array<scalar_t>(up.data_ptr<scalar_t>(), total_elements);
            std::vector<ck::ChunkedArray<scalar_t>> output_grad_chunks =
                ck::chunk_array<scalar_t>(output_grad.data_ptr<scalar_t>(), total_elements);
            std::vector<ck::ChunkedArray<scalar_t>> gate_grad_chunks =
                ck::chunk_array<scalar_t>(gate_grad.data_ptr<scalar_t>(), total_elements);
            std::vector<ck::ChunkedArray<scalar_t>> up_grad_chunks =
                ck::chunk_array<scalar_t>(up_grad.data_ptr<scalar_t>(), total_elements);

            for (int i = 0; i < gate_chunks.size(); i++) {
                ck::ChunkedArray<scalar_t> gate_chunk = gate_chunks[i];
                ck::ChunkedArray<scalar_t> up_chunk = up_chunks[i];
                ck::ChunkedArray<scalar_t> output_grad_chunk = output_grad_chunks[i];
                ck::ChunkedArray<scalar_t> gate_grad_chunk = gate_grad_chunks[i];
                ck::ChunkedArray<scalar_t> up_grad_chunk = up_grad_chunks[i];

                const uint64 num_elements = gate_chunk.num_elements;
                const uint32 NUM_BLOCKS = ck::ceil_divide<uint64>(num_elements, num_elements_per_block);

                _swiglu_backward_cuda_kernel<scalar_t><<<NUM_BLOCKS, BLOCK_SIZE>>>(gate_chunk.array,
                                                                                   up_chunk.array,
                                                                                   output_grad_chunk.array,
                                                                                   gate_grad_chunk.array,
                                                                                   up_grad_chunk.array,
                                                                                   num_elements);
            }
        }));
}
