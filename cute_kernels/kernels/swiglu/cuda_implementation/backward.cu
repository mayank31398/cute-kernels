#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "../../../include/activations.h"
#include "../../../include/dtypes/all.h"
#include "../../../include/launch.h"
#include "../../../include/math.h"
#include "../../../include/threads.h"

template <typename scalar_t>
__global__ void _swiglu_backward_cuda_kernel(const scalar_t *gate,
                                             const scalar_t *up,
                                             const scalar_t *output_grad,
                                             scalar_t *gate_grad,
                                             scalar_t *up_grad,
                                             const uint64 num_elements) {
    constexpr int vector_instruction_width = sizeof(fp32_4) / sizeof(scalar_t);
    static_assert(vector_instruction_width == 4 || vector_instruction_width == 8);

    const uint64 thread_id = get_global_thread_id();
    using dtype = DType<scalar_t>;

    uint64 end = (thread_id + 1) * vector_instruction_width - 1;  // inclusive of last element

    if (end < num_elements) {
        const fp32 *gate_vec = (fp32 *)&((fp32_4 *)gate)[thread_id];
        const fp32 *up_vec = (fp32 *)&((fp32_4 *)up)[thread_id];
        const fp32 *output_grad_vec = (fp32 *)&((fp32_4 *)output_grad)[thread_id];

        fp32 gate_grad_buffer[4];
        fp32 up_grad_buffer[4];

        // clang-format off
        #pragma unroll
        // clang-format on
        for (int i = 0; i < 4; i++) {
            if constexpr (std::is_same_v<scalar_t, fp32>) {
                fp32 _gate_sigmoid = sigmoid<fp32, fp32>(gate_vec[i]);
                fp32 _gate_silu = gate_vec[i] * _gate_sigmoid;

                gate_grad_buffer[i] =
                    output_grad_vec[i] * up_vec[i] * (_gate_sigmoid + _gate_silu * (1 - _gate_sigmoid));
                up_grad_buffer[i] = output_grad_vec[i] * _gate_silu;
            } else {
                fp32_2 _gate_upcast = dtype::upcast(dtype::reinterpret_32_bits_as_2x16(gate_vec[i]));
                fp32_2 _up_upcast = dtype::upcast(dtype::reinterpret_32_bits_as_2x16(up_vec[i]));
                fp32_2 _output_grad_upcast = dtype::upcast(dtype::reinterpret_32_bits_as_2x16(output_grad_vec[i]));

                fp32 _gate_sigmoid_x = sigmoid<fp32, fp32>(_gate_upcast.x);
                fp32 _gate_sigmoid_y = sigmoid<fp32, fp32>(_gate_upcast.y);

                fp32 _gate_silu_x = _gate_upcast.x * _gate_sigmoid_x;
                fp32 _gate_silu_y = _gate_upcast.y * _gate_sigmoid_y;

                _gate_upcast = DType<fp32>::make2(
                    _output_grad_upcast.x * _up_upcast.x * (_gate_sigmoid_x + _gate_silu_x * (1 - _gate_sigmoid_x)),
                    _output_grad_upcast.y * _up_upcast.y * (_gate_sigmoid_y + _gate_silu_y * (1 - _gate_sigmoid_y)));

                _up_upcast =
                    DType<fp32>::make2(_output_grad_upcast.x * _gate_silu_x, _output_grad_upcast.y * _gate_silu_y);

                gate_grad_buffer[i] = dtype::reinterpret_2x16_as_32_bits(dtype::downcast(_gate_upcast));
                up_grad_buffer[i] = dtype::reinterpret_2x16_as_32_bits(dtype::downcast(_up_upcast));
            }
        }

        ((fp32_4 *)gate_grad)[thread_id] = DType<fp32>::make4(gate_grad_buffer);
        ((fp32_4 *)up_grad)[thread_id] = DType<fp32>::make4(up_grad_buffer);
    }

    // use first warp for computing the last elements
    if (thread_id < WARP_SIZE) {
        // NOTE end is same as start since we don't use vector load stores here
        end = (num_elements / vector_instruction_width) * vector_instruction_width + thread_id;
        if (end < num_elements) {
            fp32 _gate_upcast = dtype::upcast(gate[end]);

            fp32 _gate_sigmoid = sigmoid<fp32, fp32>(_gate_upcast);
            fp32 _gate_silu = _gate_upcast * _gate_sigmoid;

            gate_grad[end] =
                dtype::downcast(output_grad[end] * up[end] * (_gate_sigmoid + _gate_silu * (1 - _gate_sigmoid)));
            up_grad[end] = dtype::downcast(output_grad[end] * _gate_silu);
        }
    }
}

void swiglu_backward_cuda(const torch::Tensor &gate,
                          const torch::Tensor &up,
                          const torch::Tensor &output_grad,
                          torch::Tensor &gate_grad,
                          torch::Tensor &up_grad,
                          const uint32 &BLOCK_SIZE) {
    const uint64 total_elements = gate.numel();

    AT_DISPATCH_CUSTOM_FLOAT_TYPES(
        gate.scalar_type(), "swiglu_backward_cuda_kernel", ([&] {
            const uint32 vector_instruction_width = 16 / sizeof(scalar_t);

            std::vector<ChunkedArray<scalar_t>> gate_chunks =
                chunk_array<scalar_t>(gate.data_ptr<scalar_t>(), total_elements);
            std::vector<ChunkedArray<scalar_t>> up_chunks =
                chunk_array<scalar_t>(up.data_ptr<scalar_t>(), total_elements);
            std::vector<ChunkedArray<scalar_t>> output_grad_chunks =
                chunk_array<scalar_t>(output_grad.data_ptr<scalar_t>(), total_elements);
            std::vector<ChunkedArray<scalar_t>> gate_grad_chunks =
                chunk_array<scalar_t>(gate_grad.data_ptr<scalar_t>(), total_elements);
            std::vector<ChunkedArray<scalar_t>> up_grad_chunks =
                chunk_array<scalar_t>(up_grad.data_ptr<scalar_t>(), total_elements);

            for (int i = 0; i < gate_chunks.size(); i++) {
                ChunkedArray<scalar_t> gate_chunk = gate_chunks[i];
                ChunkedArray<scalar_t> up_chunk = up_chunks[i];
                ChunkedArray<scalar_t> output_grad_chunk = output_grad_chunks[i];
                ChunkedArray<scalar_t> gate_grad_chunk = gate_grad_chunks[i];
                ChunkedArray<scalar_t> up_grad_chunk = up_grad_chunks[i];

                const uint64 num_elements = gate_chunk.num_elements;

                const uint32 num_elements_per_block = BLOCK_SIZE * vector_instruction_width;
                const uint32 NUM_BLOCKS = ceil_divide<uint64>(num_elements, num_elements_per_block);

                _swiglu_backward_cuda_kernel<scalar_t><<<NUM_BLOCKS, BLOCK_SIZE>>>(gate_chunk.array,
                                                                                   up_chunk.array,
                                                                                   output_grad_chunk.array,
                                                                                   gate_grad_chunk.array,
                                                                                   up_grad_chunk.array,
                                                                                   num_elements);
            }
        }));
}
