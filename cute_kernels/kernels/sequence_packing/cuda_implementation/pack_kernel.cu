#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "include/cute_kernels.h"

namespace ck = cute_kernels;
namespace ck_mem = ck::memory;

using fp32 = ck::fp32;
using uint32 = ck::uint32;
using uint64 = ck::uint64;

template <typename integer_t>
inline __device__ void _load_cu_seqlens(const integer_t *cu_seqlens, integer_t *cu_seqlens_shared, const uint32 &B) {
    constexpr uint32 num_elements_per_thread = sizeof(integer_t);
    const uint32 B4 = B / num_elements_per_thread;

    for (uint32 i = threadIdx.x; i < B4; i += blockDim.x) {
        const uint32 index = i * num_elements_per_thread;
        uint32 *cu_seqlens_loaded = ck_mem::load_128_bits<integer_t>(cu_seqlens, i);

        for (uint32 j = 0; j < num_elements_per_thread; j++) {
            cu_seqlens_shared[index + j] = cu_seqlens_loaded[j];
        }
    }

    // use first warp to load remaining elements
    const uint32 index = (B4 * num_elements_per_thread) + threadIdx.x;
    if (index < B) {
        cu_seqlens_shared[index] = cu_seqlens[index];
    }
}

template <typename scalar_t, typename integer_t>
__global__ void _pack_sequence_cuda_kernel(const scalar_t *x,
                                           scalar_t *output,
                                           const uint32 *cu_seqlens,
                                           const uint32 *max_seqlen_tensor,
                                           const uint32 max_seqlen,
                                           const uint32 B,
                                           const uint32 S) {
    __shared__ integer_t max_seqlen_shared;
    __shared__ integer_t cu_seqlens_shared[B];

    _load_cu_seqlens<integer_t>(cu_seqlens, cu_seqlens_shared, B);

    // load max_seqlen into shared memory using 1st thread of each threadblock
    if (threadIdx.x == 0) {
        max_seqlen_shared = max_seqlen_tensor[0];
    }

    __syncthreads();
}

void pack_sequence_cuda(const torch::Tensor &x,
                        torch::Tensor &output,
                        const torch::Tensor &cu_seqlens,
                        const std::optional<torch::Tensor> &max_seqlen_tensor,
                        const std::optional<uint32> &max_seqlen,
                        const string &padding_side,
                        const uint32 &BLOCK_SIZE) {
    CHECK_CUDA_TENSOR(x);
    CHECK_CUDA_TENSOR(output);
    CHECK_CUDA_TENSOR(cu_seqlens);
    CHECK_CUDA_TENSOR(max_seqlen_tensor);

    CHECK_VALID_THREAD_BLOCK(BLOCK_SIZE);

    TORCH_CHECK(padding_side == "left" || padding_side == "right");

    // only one of the 2 should contain a value
    if (max_seqlen_tensor.has_value()) {
        TORCH_CHECK(!max_seqlen.has_value());
    } else {
        TORCH_CHECK(max_seqlen.has_value());
    }

    const uint64 num_elements = x.numel();
    // FIXME check this value
    TORCH_CHECK(num_elements < 1000000000);

    AT_DISPATCH_CUSTOM_FLOAT_TYPES(
        x.scalar_type(), "pack_sequence_cuda_kernel", ([&] {
            constexpr uint32 num_elements_per_thread = ck_mem::get_num_elements_for_vector_load_stores<scalar_t>();
            const uint32 num_elements_per_block = num_elements_per_thread * BLOCK_SIZE;

            TORCH_CHECK(num_elements % num_elements_per_thread == 0);

            const uint32 NUM_BLOCKS = ck::ceil_divide<uint64>(num_elements, num_elements_per_block);

            _pack_sequence_cuda_kernel<scalar_t, uint32><<<NUM_BLOCKS, BLOCK_SIZE>>>(
                x.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                cu_seqlens.data_ptr<uint32>(),
                max_seqlen_tensor.has_value() ? max_seqlen_tensor.value().data_ptr<uint32>() : nullptr,
                max_seqlen.has_value() ? max_seqlen.value() : nullptr,
                B,
                S);
        }));
}
