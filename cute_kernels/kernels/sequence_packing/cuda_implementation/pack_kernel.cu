#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "include/cute_kernels.h"

namespace ck = cute_kernels;
namespace ck_mem = ck::memory;

using fp32 = ck::fp32;
using uint32 = ck::uint32;
using uint64 = ck::uint64;

template <typename integer_t>
inline __device__ void _load_cu_seqlens(const integer_t *cu_seqlens, integer_t *cu_seqlens_shared, const uint32 &B) {
    constexpr uint32 num_elements_per_thread = sizeof(integer_t);
    const uint32 B4 = B / num_elements_per_thread;

    for (uint32 i = threadIdx.x; i < B4; i += blockDim.x) {
        const uint32 index = i * num_elements_per_thread;
        uint32 *cu_seqlens_loaded = ck_mem::load_128_bits<integer_t>(cu_seqlens, i);

        for (uint32 j = 0; j < num_elements_per_thread; j++) {
            cu_seqlens_shared[index + j] = cu_seqlens_loaded[j];
        }
    }

    // use first warp to load remaining elements
    const uint32 index = (B4 * num_elements_per_thread) + threadIdx.x;
    if (index < B) {
        cu_seqlens_shared[index] = cu_seqlens[index];
    }
}

template <typename scalar_t, typename integer_t, bool is_max_seqlen_tensor>
__global__ void _pack_sequence_cuda_kernel(const scalar_t *x,
                                           scalar_t *output,
                                           const uint32 *cu_seqlens,
                                           const uint32 *max_seqlen_tensor,
                                           const uint32 max_seqlen,
                                           const uint32 B,
                                           const uint32 S) {
    __shared__ integer_t max_seqlen_shared;
    integer_t *cu_seqlens_shared = ck_mem::get_dynamic_shared_memory<integer_t>();

    // _load_cu_seqlens<integer_t>(cu_seqlens, cu_seqlens_shared, B);

    // load max_seqlen into shared memory using 1st thread of each threadblock
    if (threadIdx.x == 0) {
        if (is_max_seqlen_tensor) {
            max_seqlen_shared = max_seqlen_tensor[0];
        } else {
            max_seqlen_shared = max_seqlen;
        }
    }

    __syncthreads();
}

void pack_sequence_cuda(const torch::Tensor &x,
                        torch::Tensor &output,
                        const torch::Tensor &cu_seqlens,
                        const std::optional<torch::Tensor> &max_seqlen_tensor,
                        const std::optional<uint32> &max_seqlen,
                        const std::string &padding_side,
                        const uint32 &BLOCK_SIZE) {
    CHECK_CUDA_TENSOR(x);
    CHECK_CUDA_TENSOR(output);
    CHECK_CUDA_TENSOR(cu_seqlens);

    CHECK_VALID_THREAD_BLOCK(BLOCK_SIZE);

    TORCH_CHECK(padding_side == "left" || padding_side == "right");

    // only one of the 2 should contain a value
    if (max_seqlen_tensor.has_value()) {
        CHECK_CUDA_TENSOR(max_seqlen_tensor.value());
        TORCH_CHECK(!max_seqlen.has_value());
    } else {
        TORCH_CHECK(max_seqlen.has_value());
    }

    const uint32 B = x.size(0);
    const uint32 S = x.size(1);

    const uint64 num_elements = x.numel();
    // FIXME check this value
    TORCH_CHECK(num_elements < 1000000000);

    AT_DISPATCH_CUSTOM_FLOAT_TYPES(
        x.scalar_type(), "pack_sequence_cuda_kernel", ([&] {
            constexpr uint32 num_elements_per_thread = ck_mem::get_num_elements_for_vector_load_stores<scalar_t>();
            const uint32 num_elements_per_block = num_elements_per_thread * BLOCK_SIZE;

            TORCH_CHECK(num_elements % num_elements_per_thread == 0);

            const uint32 NUM_BLOCKS = ck::ceil_divide<uint64>(num_elements, num_elements_per_block);
            const uint32 shared_memory_size = (cu_seqlens.numel() + 1) * sizeof(uint32);

            if (max_seqlen_tensor.has_value()) {
                _pack_sequence_cuda_kernel<scalar_t, uint32, true>
                    <<<NUM_BLOCKS, BLOCK_SIZE, shared_memory_size>>>(x.data_ptr<scalar_t>(),
                                                                     output.data_ptr<scalar_t>(),
                                                                     cu_seqlens.data_ptr<uint32>(),
                                                                     max_seqlen_tensor.value().data_ptr<uint32>(),
                                                                     0,
                                                                     B,
                                                                     S);
            } else {
                _pack_sequence_cuda_kernel<scalar_t, uint32, false>
                    <<<NUM_BLOCKS, BLOCK_SIZE, shared_memory_size>>>(x.data_ptr<scalar_t>(),
                                                                     output.data_ptr<scalar_t>(),
                                                                     cu_seqlens.data_ptr<uint32>(),
                                                                     nullptr,
                                                                     max_seqlen.value(),
                                                                     B,
                                                                     S);
            }
        }));
}
