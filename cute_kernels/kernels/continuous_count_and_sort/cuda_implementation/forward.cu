#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "../../../include/dtypes/all.h"
#include "../../../include/launch.h"
#include "../../../include/math.h"
#include "../../../include/threads.h"

#define MAX_ALLOWED_C 16384

namespace cg = cooperative_groups;

inline __device__ void _looped_atomic_add(uint32 *output_shared,
                                          uint32 *destination_output_shared,
                                          const uint32 &num_loops_C,
                                          const uint32 &C,
                                          const uint32 &local_thread_id) {
    for (int i = 0; i < num_loops_C; i++) {
        const int index = i * blockDim.x + local_thread_id;
        if (index < C) {
            atomicAdd(&destination_output_shared[index], output_shared[index]);
        }
    }
}

template <typename scalar_t>
__global__ void _continuous_count_and_sort_cuda_kernel(const scalar_t *x,
                                                       uint32 *count,
                                                       const uint64 num_elements,
                                                       const uint32 C) {
    const uint32 local_thread_id = get_local_thread_id();
    const uint32 num_loops_C = ceil_divide<uint32>(C, blockDim.x);

    extern __shared__ uint32 output_shared[];

    for (uint32 i = 0; i < num_loops_C; i++) {
        const uint32 index = i * blockDim.x + local_thread_id;
        if (index < C) {
            count[index] = 0;
            output_shared[index] = 0;
        }
    }

    __syncthreads();

    // count the number of occurances of each number in x
    const uint32 num_elements_per_block = ceil_divide<uint64>(num_elements, gridDim.x);

    const uint32 start = blockIdx.x * num_elements_per_block;
    uint64 end = start + num_elements_per_block;
    if (end > num_elements) {
        end = num_elements;
    }

    const int num_elements_in_current_block = end - start;

    if (num_elements_in_current_block > 0) {
        const uint32 num_loops = ceil_divide<uint32>(num_elements_in_current_block, blockDim.x);

        for (int i = 0; i < num_loops; i++) {
            const int index = start + i * blockDim.x + local_thread_id;
            if (index < end) {
                atomicAdd(&output_shared[x[index]], 1);
            }
        }

        __syncthreads();

        for (int i = 0; i < num_loops_C; i++) {
            const int index = i * blockDim.x + local_thread_id;
            if (index < C) {
                atomicAdd(&count[index], output_shared[index]);
            }
        }
    }
}

void continuous_count_and_sort_cuda(
    const torch::Tensor &x, torch::Tensor &count, const uint32 &sm_count, const uint32 &C, const uint32 &BLOCK_SIZE) {
    assert(BLOCK_SIZE % WARP_SIZE == 0);
    assert(C <= MAX_ALLOWED_C);

    const uint64 total_elements = x.numel();

    std::vector<ChunkedArray<uint32>> count_chunks = chunk_array<uint32>(count.data_ptr<uint32>(), total_elements);

    AT_DISPATCH_CUSTOM_INT_TYPES(x.scalar_type(), "continuous_count_and_sort_cuda_kernel", ([&] {
                                     hipFuncSetAttribute(reinterpret_cast<const void*>(_continuous_count_cuda_kernel<scalar_t>),
                                                          hipFuncAttributeMaxDynamicSharedMemorySize,
                                                          MAX_ALLOWED_C * sizeof(uint32));

                                     std::vector<ChunkedArray<scalar_t>> x_chunks =
                                         chunk_array<scalar_t>(x.data_ptr<scalar_t>(), total_elements);

                                     for (int i = 0; i < x_chunks.size(); i++) {
                                         ChunkedArray<scalar_t> x_chunk = x_chunks[i];
                                         ChunkedArray<uint32> count_chunk = count_chunks[i];

                                         const uint32 num_elements = x_chunk.num_elements;

                                         auto [NUM_BLOCKS, cluster_size] = get_num_blocks(
                                             num_elements, BLOCK_SIZE, sm_count, thread_block_cluster_size);

                                         _continuous_count_and_sort_cuda_kernel<scalar_t><<<NUM_BLOCKS, BLOCK_SIZE>>>(
                                             x_chunk.array, count_chunk.array, num_elements, C);
                                     }
                                 }));
}
