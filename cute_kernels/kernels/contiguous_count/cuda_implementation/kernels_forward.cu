#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "../../../include/dtypes/all.h"
#include "../../../include/threads.h"

#define MAX_ALLOWED_C 16384

inline __device__ void _init_shared_memory(uint32 *x, const int &C, const uint32 &thread_id) {
    const int num_loops = (C + blockDim.x - 1) / blockDim.x;
    // clang-format off
    #pragma unroll
    // clang-format on
    for (int i = 0; i < num_loops; i++) {
        x[thread_id + i * blockDim.x] = 0;
    }
}

template <typename scalar_t, int vector_instruction_width>
__global__ void _contiguous_count_cuda_kernel(const scalar_t *x,
                                              const scalar_t *output,
                                              const uint64 num_elements,
                                              const uint32 C) {
    const uint64 thread_id = get_global_thread_id();

    __shared__ uint32 output_shared[MAX_ALLOWED_C];
    _init_shared_memory(output_shared, C, thread_id);
    __syncthreads();

    const uint32 *x_vec = (uint32 *)&((uint32_4 *)x)[thread_id];

    // clang-format off
    #pragma unroll
    // clang-format on
    for (int i = 0; i < 4; i++) {
        uint32 *x_local = (uint32 *)x_vec[thread_id];
        x_local[i];
    }
}

void contiguous_count_cuda(const torch::Tensor &x, const torch::Tensor &output, const int &C, const int &BLOCK_SIZE) {
    assert(BLOCK_SIZE % WARP_SIZE == 0);
    assert(C < MAX_ALLOWED_C);

    const uint64 num_elements = x.numel();

    // we use vector instructions of width 4
    const int num_elements_per_block = BLOCK_SIZE << 2;
    const int NUM_BLOCKS = (num_elements + num_elements_per_block - 1) / num_elements_per_block;

    AT_DISPATCH_CUSTOM_INT_TYPES(x.scalar_type(), "contiguous_count_cuda_kernel", ([&] {
                                     _contiguous_count_cuda_kernel<scalar_t, 4><<<NUM_BLOCKS, BLOCK_SIZE>>>(
                                         x.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(), num_elements, C);
                                 }));
}
