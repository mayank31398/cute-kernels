#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "../../../include/dtypes/all.h"
#include "../../../include/threads.h"

#define MAX_ALLOWED_C 16384

template <int vector_instruction_width>
__global__ void _contiguous_count_cuda_kernel(const uint32 *x,
                                              const uint32 *output,
                                              const uint64 num_elements,
                                              const uint32 C) {
    __shared__ uint32 output_shared[MAX_ALLOWED_C];

    const uint64 thread_id = get_global_thread_id();
    const uint32 *x_vec = (uint32 *)&((uint32_4 *)x)[thread_id];

    // clang-format off
    #pragma unroll
    // clang-format on
    for (int i = 0; i < 4; i++) {
        uint32 *x_local = (uint32 *)x_vec[thread_id];
        x_local[i];
    }
}

void contiguous_count_cuda(const torch::Tensor &x, const torch::Tensor &output, const int &C, const int &BLOCK_SIZE) {
    assert(BLOCK_SIZE % WARP_SIZE == 0);
    assert(C < MAX_ALLOWED_C);

    const uint64 num_elements = x.numel();

    // we use vector instructions of width 4
    const int num_elements_per_block = BLOCK_SIZE << 2;
    const int NUM_BLOCKS = (num_elements + num_elements_per_block - 1) / num_elements_per_block;

    // the 3rd argument in launch parameters is the size of dynamic shared memory
    _contiguous_count_cuda_kernel<4>
        <<<NUM_BLOCKS, BLOCK_SIZE>>>(x.data_ptr<uint>(), output.data_ptr<uint>(), num_elements, C);
}
