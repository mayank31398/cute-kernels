#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "../../../include/device.h"
#include "../../../include/dtypes/all.h"
#include "../../../include/threads.h"

#define MAX_ALLOWED_C 16384

__global__ void _contiguous_count_cuda_kernel(const int32 *x,
                                              int32 *output,
                                              const uint64 num_elements,
                                              const uint32 C) {
    const uint64 thread_id = get_global_thread_id();
    const int num_loops_C = (C + blockDim.x - 1) / blockDim.x;

    extern __shared__ uint32 output_shared[];

    // initialize shared memory and output
    // clang-format off
    #pragma unroll
    // clang-format on
    for (int i = 0; i < num_loops_C; i++) {
        const int index = thread_id + i * blockDim.x;
        if (index < C) {
            output_shared[index] = 0;
            output[index] = 0;
        }
    }

    __syncthreads();

    // count the number of occurances of each number in x
    const int num_elements_per_block = (num_elements + gridDim.x - 1) / gridDim.x;
    const int num_loops_B = (num_elements_per_block + blockDim.x - 1) / blockDim.x;

    const int start = blockIdx.x * num_elements_per_block;

    for (int i = 0; i < num_loops_B; i++) {
        const int index = start + i * num_loops_B + thread_id;
        if (index < num_elements) {
            atomicAdd(&output_shared[x[index]], 1);
        }
    }

    __syncthreads();

    // write the output to the global memory
    // clang-format off
    #pragma unroll
    // clang-format on
    for (int i = 0; i < num_loops_C; i++) {
        const int index = thread_id + i * blockDim.x;
        if (index < C) {
            atomicAdd(&output[index], output_shared[index]);
        }
    }
}

void contiguous_count_cuda(const torch::Tensor &x, const torch::Tensor &output, const int &C, const int &BLOCK_SIZE) {
    assert(BLOCK_SIZE % WARP_SIZE == 0);
    assert(C < MAX_ALLOWED_C);

    const uint64 num_elements = x.numel();

    // we use vector instructions of width 4
    const int num_elements_per_block = BLOCK_SIZE << 2;
    const int NUM_BLOCKS = (num_elements + num_elements_per_block - 1) / num_elements_per_block;

    _contiguous_count_cuda_kernel<<<NUM_BLOCKS, BLOCK_SIZE, C * sizeof(int32)>>>(
        x.data_ptr<int32>(), output.data_ptr<int32>(), num_elements, C);
}
