#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "../../../include/dtypes/all.h"
#include "../../../include/threads.h"

#define MAX_ALLOWED_C 16384

template <int vector_instruction_width>
__global__ void _contiguous_count_cuda_kernel(const int32 *x,
                                              int32 *output,
                                              const uint64 num_elements,
                                              const uint32 C) {
    const uint64 thread_id = get_global_thread_id();
    const int num_loops = (C + blockDim.x - 1) / blockDim.x;

    __shared__ uint32 output_shared[MAX_ALLOWED_C];

    // clang-format off
    #pragma unroll
    // clang-format on
    for (int i = 0; i < num_loops; i++) {
        const int index = thread_id + i * blockDim.x;
        if (index < C) {
            output_shared[index] = 0;
            output[index] = 3;
        }
    }

    // __syncthreads();

    // // TODO add code here

    // __syncthreads();

    // // clang-format off
    // #pragma unroll
    // // clang-format on
    // for (int i = 0; i < num_loops; i++) {
    //     const int index = thread_id + i * blockDim.x;
    //     if (index < C) {
    //         atomicAdd(&output[index], output_shared[index]);
    //     }
    // }
}

void contiguous_count_cuda(const torch::Tensor &x, const torch::Tensor &output, const int &C, const int &BLOCK_SIZE) {
    assert(BLOCK_SIZE % WARP_SIZE == 0);
    assert(C < MAX_ALLOWED_C);

    const uint64 num_elements = x.numel();

    // we use vector instructions of width 4
    const int num_elements_per_block = BLOCK_SIZE << 2;
    const int NUM_BLOCKS = (num_elements + num_elements_per_block - 1) / num_elements_per_block;

    _contiguous_count_cuda_kernel<4>
        <<<NUM_BLOCKS, BLOCK_SIZE>>>(x.data_ptr<int32>(), output.data_ptr<int32>(), num_elements, C);
}
