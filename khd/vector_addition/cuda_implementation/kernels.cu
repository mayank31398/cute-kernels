#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#define BLOCK_SIZE 1024

// for vectorized load store
#define NUM_ELEMENTS_PER_THREAD_FP32 4
#define NUM_ELEMENTS_PER_THREAD_FP16 8
#define NUM_ELEMENTS_PER_THREAD_BF16 8

__device__ half2 unpack(float value) {
    uint32_t intValue = __float_as_int(value);

    uint16_t lower16Bits = intValue & 0xFFFF;
    uint16_t higher16Bits = (intValue >> 16) & 0xFFFF;

    half lowerHalf = __ushort_as_half(lower16Bits);
    half higherHalf = __ushort_as_half(higher16Bits);

    return __halves2half2(lowerHalf, higherHalf);
}

__device__ float pack(half2 value) {
    // Extract the two half values from half2
    half lowerHalf = __low2half(value);
    half higherHalf = __high2half(value);

    // Convert the half values to uint16_t
    uint16_t lower16Bits = __half_as_short(lowerHalf);
    uint16_t higher16Bits = __half_as_short(higherHalf);

    // Combine the 16-bit values into a single 32-bit integer
    uint32_t intValue = (static_cast<uint32_t>(higher16Bits) << 16) | lower16Bits;

    // Convert the 32-bit integer back to float
    return __int_as_float(intValue);
}

template <typename scalar_t>
__global__ void vector_addition_forward_kernel(const scalar_t *x,
                                               const scalar_t *y,
                                               scalar_t *output,
                                               const int num_elements,
                                               const int num_elements_per_thread) {
    const int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    const int start = thread_id * num_elements_per_thread;
    const int end = (thread_id + 1) * num_elements_per_thread - 1; // inclusive of last element

    if (start < num_elements && end < num_elements) {
        // float4 is a datatype used for vectorized loads and stores
        const float4 *x4 = (const float4 *)x;
        const float4 *y4 = (const float4 *)y;
        float4 *output4 = (float4 *)output;

        // tmp is initialized here to avoid doing multiple writes
        const float4 _x4 = x4[thread_id];
        const float4 _y4 = y4[thread_id];
        float4 tmp;

        if (std::is_same_v<scalar_t, float>) {
            tmp.x = _x4.x + _y4.x;
            tmp.y = _x4.y + _y4.y;
            tmp.z = _x4.z + _y4.z;
            tmp.w = _x4.w + _y4.w;
        } else if (std::is_same_v<scalar_t, c10::Half>) {
            tmp.x = pack(unpack(_x4.x) + unpack(_y4.x));
            tmp.y = pack(unpack(_x4.y) + unpack(_y4.y));
            tmp.z = pack(unpack(_x4.z) + unpack(_y4.z));
            tmp.w = pack(unpack(_x4.w) + unpack(_y4.w));
        } else if (std::is_same_v<scalar_t, c10::BFloat16>) {
            tmp.x = pack(unpack(_x4.x) + unpack(_y4.x));
            tmp.y = pack(unpack(_x4.y) + unpack(_y4.y));
            tmp.z = pack(unpack(_x4.z) + unpack(_y4.z));
            tmp.w = pack(unpack(_x4.w) + unpack(_y4.w));
        }

        output4[thread_id] = tmp;
    } else if (start < num_elements) {
#pragma unroll
        for (int i = start; i < num_elements; i++) {
            output[i] = x[i] + y[i];
        }
    }
}

torch::Tensor vector_addition_forward_kernel_dispatcher(torch::Tensor x, torch::Tensor y) {
    int num_elements = x.numel();

    torch::Tensor output = torch::empty_like(x);

    AT_DISPATCH_FLOATING_TYPES_AND2(
        at::ScalarType::Half, at::ScalarType::BFloat16, x.scalar_type(), "vector_addition_forward_kernel", ([&] {
            int num_elements_per_thread;
            if (std::is_same_v<scalar_t, float>) {
                num_elements_per_thread = NUM_ELEMENTS_PER_THREAD_FP32;
            } else if (std::is_same_v<scalar_t, c10::Half>) {
                num_elements_per_thread = NUM_ELEMENTS_PER_THREAD_FP16;
            } else if (std::is_same_v<scalar_t, c10::BFloat16>) {
                num_elements_per_thread = NUM_ELEMENTS_PER_THREAD_BF16;
            }

            int num_elements_per_block = BLOCK_SIZE * num_elements_per_thread;
            int NUM_BLOCKS = (num_elements + num_elements_per_block - 1) / num_elements_per_block;

            vector_addition_forward_kernel<scalar_t><<<NUM_BLOCKS, BLOCK_SIZE>>>(x.data<scalar_t>(),
                                                                                 y.data<scalar_t>(),
                                                                                 output.data<scalar_t>(),
                                                                                 num_elements,
                                                                                 num_elements_per_thread);
        }));

    return output;
}
