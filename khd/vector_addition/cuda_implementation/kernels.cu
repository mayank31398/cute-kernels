#include "hip/hip_runtime.h"
#include "../../dtypes.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#define BLOCK_SIZE 1024

// for vectorized load store
#define NUM_ELEMENTS_PER_THREAD_FP32 4
#define NUM_ELEMENTS_PER_THREAD_FP16 8
#define NUM_ELEMENTS_PER_THREAD_BF16 8

template <typename scalar_t>
__global__ void vector_addition_forward_kernel(const scalar_t *x,
                                               const scalar_t *y,
                                               scalar_t *output,
                                               const int num_elements,
                                               const int num_elements_per_thread) {
    const int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    const int start = thread_id * num_elements_per_thread;
    const int end = (thread_id + 1) * num_elements_per_thread - 1; // inclusive of last element

    if (start < num_elements && end < num_elements) {
        // fp32_4 is a datatype used for vectorized loads and stores
        const fp32_4 *x4 = (const fp32_4 *)x;
        const fp32_4 *y4 = (const fp32_4 *)y;
        fp32_4 *output4 = (fp32_4 *)output;

        // tmp is initialized here to avoid doing multiple writes
        const fp32_4 _x4 = x4[thread_id];
        const fp32_4 _y4 = y4[thread_id];
        fp32_4 tmp;

        if (std::is_same_v<scalar_t, fp32>) {
            tmp.x = _x4.x + _y4.x;
            tmp.y = _x4.y + _y4.y;
            tmp.z = _x4.z + _y4.z;
            tmp.w = _x4.w + _y4.w;
        } else if (std::is_same_v<scalar_t, c10::Half>) {
            DType<c10::Half> q;
            tmp.x = q.pack(__hadd2(q.unpack(_x4.x), q.unpack(_y4.x)));
            tmp.y = q.pack(__hadd2(q.unpack(_x4.y), q.unpack(_y4.y)));
            tmp.z = q.pack(__hadd2(q.unpack(_x4.z), q.unpack(_y4.z)));
            tmp.w = q.pack(__hadd2(q.unpack(_x4.w), q.unpack(_y4.w)));
        } else if (std::is_same_v<scalar_t, c10::BFloat16>) {
            DType<c10::BFloat16> q;
            tmp.x = q.pack(__hadd2(q.unpack(_x4.x), q.unpack(_y4.x)));
            tmp.y = q.pack(__hadd2(q.unpack(_x4.y), q.unpack(_y4.y)));
            tmp.z = q.pack(__hadd2(q.unpack(_x4.z), q.unpack(_y4.z)));
            tmp.w = q.pack(__hadd2(q.unpack(_x4.w), q.unpack(_y4.w)));
        }

        output4[thread_id] = tmp;
    } else if (start < num_elements) {
#pragma unroll
        for (int i = start; i < num_elements; i++) {
            output[i] = x[i] + y[i];
        }
    }
}

torch::Tensor vector_addition_forward_kernel_dispatcher(torch::Tensor x, torch::Tensor y) {
    int num_elements = x.numel();

    torch::Tensor output = torch::empty_like(x);

    AT_DISPATCH_FLOATING_TYPES_AND2(
        at::ScalarType::Half, at::ScalarType::BFloat16, x.scalar_type(), "vector_addition_forward_kernel", ([&] {
            int num_elements_per_thread;
            if (std::is_same_v<scalar_t, float>) {
                num_elements_per_thread = NUM_ELEMENTS_PER_THREAD_FP32;
            } else if (std::is_same_v<scalar_t, c10::Half>) {
                num_elements_per_thread = NUM_ELEMENTS_PER_THREAD_FP16;
            } else if (std::is_same_v<scalar_t, c10::BFloat16>) {
                num_elements_per_thread = NUM_ELEMENTS_PER_THREAD_BF16;
            }

            int num_elements_per_block = BLOCK_SIZE * num_elements_per_thread;
            int NUM_BLOCKS = (num_elements + num_elements_per_block - 1) / num_elements_per_block;

            vector_addition_forward_kernel<scalar_t><<<NUM_BLOCKS, BLOCK_SIZE>>>(x.data_ptr<scalar_t>(),
                                                                                 y.data_ptr<scalar_t>(),
                                                                                 output.data_ptr<scalar_t>(),
                                                                                 num_elements,
                                                                                 num_elements_per_thread);
        }));

    return output;
}
