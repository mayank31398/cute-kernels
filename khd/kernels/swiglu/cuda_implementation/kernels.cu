#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "../../utils/activations.h"
#include "../../utils/dtypes.h"
#include "../../utils/threads.h"

template <typename scalar_t>
__global__ void _swiglu_forward_cuda_kernel(const scalar_t *gate,
                                            const scalar_t *up,
                                            scalar_t *output,
                                            const int64_t num_elements) {
    const int64_t thread_id = get_global_thread_id();
    const int vector_instruction_width = sizeof(fp32_4) / sizeof(scalar_t);

    const int64_t start = thread_id * vector_instruction_width;
    const int64_t end = (thread_id + 1) * vector_instruction_width - 1;  // inclusive of last element

    using dtype = DType<scalar_t>;
    using T = typename dtype::nv_dtype;

    if (start < num_elements && end < num_elements) {
        const fp32 *_gate = (fp32 *)&((fp32_4 *)gate)[thread_id];
        const fp32 *_up = (fp32 *)&((fp32_4 *)up)[thread_id];

        fp32 output_buffer[4];

        // clang-format off
        #pragma unroll
        // clang-format on
        for (int i = 0; i < 4; i++) {
            if constexpr (std::is_same_v<scalar_t, fp32>) {
                output_buffer[i] = _up[i] * _gate[i] * sigmoid<fp32, fp32>(_gate[i]);
            } else {
                fp32_2 _up_upcast = dtype::upcast(dtype::reinterpret_32_bits_as_2x16(_up[i]));
                fp32_2 _gate_upcast = dtype::upcast(dtype::reinterpret_32_bits_as_2x16(_gate[i]));

                _gate_upcast = DType<fp32>::make2(_up_upcast.x * _gate_upcast.x * sigmoid<fp32, fp32>(_gate_upcast.x),
                                                  _up_upcast.y * _gate_upcast.y * sigmoid<fp32, fp32>(_gate_upcast.y));
                output_buffer[i] = dtype::reinterpret_2x16_as_32_bits(dtype::downcast(_gate_upcast));
            }
        }

        ((fp32_4 *)output)[thread_id] = DType<fp32>::make4(output_buffer);
    } else if (start < num_elements) {
        // clang-format off
        #pragma unroll
        // clang-format on
        for (int64_t i = start; i < num_elements; i++) {
            fp32 _gate = dtype::upcast(static_cast<T>(gate[i]));
            output[i] = dtype::downcast(dtype::upcast(static_cast<T>(up[i])) * _gate * sigmoid<fp32, fp32>(_gate));
        }
    }
}

template <typename scalar_t>
__global__ void _swiglu_backward_cuda_kernel(const scalar_t *gate,
                                             const scalar_t *up,
                                             const scalar_t *output_grad,
                                             scalar_t *gate_grad,
                                             scalar_t *up_grad,
                                             const int64_t num_elements) {
    const int64_t thread_id = get_global_thread_id();
    const int vector_instruction_width = sizeof(fp32_4) / sizeof(scalar_t);

    const int64_t start = thread_id * vector_instruction_width;
    const int64_t end = (thread_id + 1) * vector_instruction_width - 1;  // inclusive of last element

    using dtype = DType<scalar_t>;
    using T = typename dtype::nv_dtype;
    using T2 = typename dtype::nv_dtype2;

    if (start < num_elements && end < num_elements) {
        const fp32 *_gate = (fp32 *)&((const fp32_4 *)gate)[thread_id];
        const fp32 *_up = (fp32 *)&((const fp32_4 *)up)[thread_id];
        const fp32 *_output_grad = (fp32 *)&((const fp32_4 *)output_grad)[thread_id];

        fp32 gate_grad_buffer[4];
        fp32 up_grad_buffer[4];

        // clang-format off
        #pragma unroll
        // clang-format on
        for (int i = 0; i < 4; i++) {
            if constexpr (std::is_same_v<scalar_t, fp32>) {
                fp32 gate_sigmoid = sigmoid<fp32, fp32>(_gate[i]);
                fp32 gate_silu = _gate[i] * gate_sigmoid;

                up_grad_buffer[i] = _output_grad[i] * gate_silu;
                gate_grad_buffer[i] = _output_grad[i] * _up[i] * (gate_sigmoid + gate_silu * (1 - gate_sigmoid));
            } else {
                fp32_2 _up_upcast = dtype::upcast(dtype::reinterpret_32_bits_as_2x16(_up[i]));
                fp32_2 _gate_upcast = dtype::upcast(dtype::reinterpret_32_bits_as_2x16(_gate[i]));
                fp32_2 _output_grad_upcast = dtype::upcast(dtype::reinterpret_32_bits_as_2x16(_output_grad[i]));

                fp32_2 _gate_grad;
                fp32_2 _up_grad;

                fp32 gate_sigmoid = sigmoid<fp32, fp32>(_gate_upcast.x);
                fp32 gate_silu = _gate_upcast.x * gate_sigmoid;
                _up_grad.x = _output_grad_upcast.x * gate_silu;
                _gate_grad.x = _output_grad_upcast.x * _up_upcast.x * (gate_sigmoid + gate_silu * (1 - gate_sigmoid));

                gate_sigmoid = sigmoid<fp32, fp32>(_gate_upcast.y);
                gate_silu = _gate_upcast.y * gate_sigmoid;
                _up_grad.y = _output_grad_upcast.y * gate_silu;
                _gate_grad.y = _output_grad_upcast.y * _up_upcast.y * (gate_sigmoid + gate_silu * (1 - gate_sigmoid));

                up_grad_buffer[i] = dtype::reinterpret_2x16_as_32_bits(dtype::downcast(_up_grad));
                gate_grad_buffer[i] = dtype::reinterpret_2x16_as_32_bits(dtype::downcast(_gate_grad));
            }
        }

        ((fp32_4 *)gate_grad)[thread_id] = DType<fp32>::make4(gate_grad_buffer);
        ((fp32_4 *)up_grad)[thread_id] = DType<fp32>::make4(up_grad_buffer);
    } else if (start < num_elements) {
        // clang-format off
        #pragma unroll
        // clang-format on
        for (int i = start; i < num_elements; i++) {
            fp32 _up = dtype::upcast(static_cast<T>(up[i]));
            fp32 _gate = dtype::upcast(static_cast<T>(gate[i]));
            fp32 _output_grad = dtype::upcast(static_cast<T>(output_grad[i]));

            fp32 gate_sigmoid = sigmoid<fp32, fp32>(_gate);
            fp32 gate_silu = _gate * gate_sigmoid;

            up_grad[i] = _output_grad * gate_silu;
            gate_grad[i] = _output_grad * _up * (gate_sigmoid + gate_silu * (1 - gate_sigmoid));
        }
    }
}

void swiglu_forward_cuda(torch::Tensor gate, torch::Tensor up, torch::Tensor output, const int BLOCK_SIZE) {
    const int64_t num_elements = gate.numel();

    AT_DISPATCH_CUSTOM_FLOAT_TYPES(
        gate.scalar_type(), "swiglu_forward_cuda_kernel", ([&] {
            const int vector_instruction_width = sizeof(fp32_4) / sizeof(scalar_t);

            const int num_elements_per_block = BLOCK_SIZE * vector_instruction_width;
            const int NUM_BLOCKS = (num_elements + num_elements_per_block - 1) / num_elements_per_block;

            _swiglu_forward_cuda_kernel<scalar_t><<<NUM_BLOCKS, BLOCK_SIZE>>>(
                gate.data_ptr<scalar_t>(), up.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(), num_elements);
        }));
}

void swiglu_backward_cuda(torch::Tensor gate,
                          torch::Tensor up,
                          torch::Tensor output_grad,
                          torch::Tensor gate_grad,
                          torch::Tensor up_grad,
                          const int BLOCK_SIZE) {
    const int64_t num_elements = gate.numel();

    AT_DISPATCH_CUSTOM_FLOAT_TYPES(
        gate.scalar_type(), "swiglu_backward_cuda_kernel", ([&] {
            const int vector_instruction_width = sizeof(fp32_4) / sizeof(scalar_t);

            const int num_elements_per_block = BLOCK_SIZE * vector_instruction_width;
            const int NUM_BLOCKS = (num_elements + num_elements_per_block - 1) / num_elements_per_block;

            _swiglu_backward_cuda_kernel<scalar_t><<<NUM_BLOCKS, BLOCK_SIZE>>>(gate.data_ptr<scalar_t>(),
                                                                               up.data_ptr<scalar_t>(),
                                                                               output_grad.data_ptr<scalar_t>(),
                                                                               gate_grad.data_ptr<scalar_t>(),
                                                                               up_grad.data_ptr<scalar_t>(),
                                                                               num_elements);
        }));
}
