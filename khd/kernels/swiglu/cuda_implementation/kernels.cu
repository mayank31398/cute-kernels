#include "hip/hip_runtime.h"
#include "../../utils/activations.h"
#include "../../utils/dtypes.h"
#include "../../utils/threads.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

template <typename scalar_t>
__global__ void _swiglu_forward_cuda_kernel(const scalar_t *gate,
                                            const scalar_t *up,
                                            scalar_t *output,
                                            const int num_elements) {
    const int thread_id = get_global_thread_id();
    const int num_elements_per_thread = get_num_elements_in_vector_dtype<scalar_t, fp32_4>();

    const int start = thread_id * num_elements_per_thread;
    const int end = (thread_id + 1) * num_elements_per_thread - 1; // inclusive of last element

    using dtype = DType<scalar_t>;
    using T = typename dtype::nv_dtype;
    using T2 = typename dtype::nv_dtype2;

    if (start < num_elements && end < num_elements) {
        const fp32 *gate_vec = (fp32 *)&((const fp32_4 *)gate)[thread_id];
        const fp32 *up_vec = (fp32 *)&((const fp32_4 *)up)[thread_id];

        fp32 output_buffer[4];

        // clang-format off
        #pragma unroll
        // clang-format on
        for (int i = 0; i < 4; i++) {
            if (std::is_same_v<scalar_t, fp32>) {
                output_buffer[i] = up_vec[i] * gate_vec[i] * sigmoid<fp32, fp32>(gate_vec[i]);
            } else if constexpr (std::is_same_v<scalar_t, c10::Half> || std::is_same_v<scalar_t, c10::BFloat16>) {
                fp32_2 _up = dtype::upcast(dtype::reinterpret_32_bits_as_2x16(up_vec[i]));
                fp32_2 _gate = dtype::upcast(dtype::reinterpret_32_bits_as_2x16(gate_vec[i]));

                _gate = DType<fp32>::make2(_up.x * _gate.x * sigmoid<fp32, fp32>(_gate.x),
                                           _up.y * _gate.y * sigmoid<fp32, fp32>(_gate.y));
                output_buffer[i] = dtype::reinterpret_2x16_as_32_bits(dtype::downcast(_gate));
            } else {
                assert(false && "Function not implemented");
            }
        }

        ((fp32_4 *)output)[thread_id] =
            make_float4(output_buffer[0], output_buffer[1], output_buffer[2], output_buffer[3]);
    } else if (start < num_elements) {
        // clang-format off
        #pragma unroll
        // clang-format on
        for (int i = start; i < num_elements; i++) {
            fp32 _gate = dtype::upcast(static_cast<T>(gate[i]));
            output[i] = dtype::downcast(dtype::upcast(static_cast<T>(up[i])) * _gate * sigmoid<fp32, fp32>(_gate));
        }
    }
}

template <typename scalar_t>
__global__ void _swiglu_backward_cuda_kernel(const scalar_t *gate,
                                             const scalar_t *up,
                                             const scalar_t *output_grad,
                                             scalar_t *gate_grad,
                                             scalar_t *up_grad,
                                             const int num_elements) {
    const int thread_id = get_global_thread_id();
    const int num_elements_per_thread = get_num_elements_in_vector_dtype<scalar_t, fp32_4>();

    const int start = thread_id * num_elements_per_thread;
    const int end = (thread_id + 1) * num_elements_per_thread - 1; // inclusive of last element

    using dtype = DType<scalar_t>;
    using T = typename dtype::nv_dtype;
    using T2 = typename dtype::nv_dtype2;

    if (start < num_elements && end < num_elements) {
        const fp32 *gate_vec = (fp32 *)&((const fp32_4 *)gate)[thread_id];
        const fp32 *up_vec = (fp32 *)&((const fp32_4 *)up)[thread_id];
        const fp32 *output_grad_vec = (fp32 *)&((const fp32_4 *)output_grad)[thread_id];

        fp32 gate_grad_buffer[4];
        fp32 up_grad_buffer[4];

        // clang-format off
        #pragma unroll
        // clang-format on
        for (int i = 0; i < 4; i++) {
            if (std::is_same_v<scalar_t, fp32>) {
                fp32 gate_sigmoid = sigmoid<fp32, fp32>(gate_vec[i]);
                fp32 gate_silu = gate_vec[i] * gate_sigmoid;

                up_grad_buffer[i] = output_grad_vec[i] * gate_silu;
                gate_grad_buffer[i] = output_grad_vec[i] * up_vec[i] * (gate_sigmoid + gate_silu * (1 - gate_sigmoid));
            } else if constexpr (std::is_same_v<scalar_t, c10::Half> || std::is_same_v<scalar_t, c10::BFloat16>) {
                fp32_2 _up = dtype::upcast(dtype::reinterpret_32_bits_as_2x16(up_vec[i]));
                fp32_2 _gate = dtype::upcast(dtype::reinterpret_32_bits_as_2x16(gate_vec[i]));
                fp32_2 _output_grad = dtype::upcast(dtype::reinterpret_32_bits_as_2x16(output_grad_vec[i]));

                fp32_2 _gate_grad;
                fp32_2 _up_grad;

                fp32 gate_sigmoid = sigmoid<fp32, fp32>(_gate.x);
                fp32 gate_silu = _gate.x * gate_sigmoid;
                _up_grad.x = _output_grad.x * gate_silu;
                _gate_grad.x = _output_grad.x * _up.x * (gate_sigmoid + gate_silu * (1 - gate_sigmoid));

                gate_sigmoid = sigmoid<fp32, fp32>(_gate.y);
                gate_silu = _gate.y * gate_sigmoid;
                _up_grad.y = _output_grad.y * gate_silu;
                _gate_grad.y = _output_grad.y * _up.y * (gate_sigmoid + gate_silu * (1 - gate_sigmoid));

                up_grad_buffer[i] = dtype::reinterpret_2x16_as_32_bits(dtype::downcast(_up_grad));
                gate_grad_buffer[i] = dtype::reinterpret_2x16_as_32_bits(dtype::downcast(_gate_grad));
            } else {
                assert(false && "Function not implemented");
            }
        }

        ((fp32_4 *)gate_grad)[thread_id] =
            make_float4(gate_grad_buffer[0], gate_grad_buffer[1], gate_grad_buffer[2], gate_grad_buffer[3]);
        ((fp32_4 *)up_grad)[thread_id] =
            make_float4(up_grad_buffer[0], up_grad_buffer[1], up_grad_buffer[2], up_grad_buffer[3]);
    } else if (start < num_elements) {
        // clang-format off
        #pragma unroll
        // clang-format on
        for (int i = start; i < num_elements; i++) {
            fp32 _up = dtype::upcast(static_cast<T>(up[i]));
            fp32 _gate = dtype::upcast(static_cast<T>(gate[i]));
            fp32 _output_grad = dtype::upcast(static_cast<T>(output_grad[i]));

            fp32 gate_sigmoid = sigmoid<fp32, fp32>(_gate);
            fp32 gate_silu = _gate * gate_sigmoid;

            up_grad[i] = _output_grad * gate_silu;
            gate_grad[i] = _output_grad * _up * (gate_sigmoid + gate_silu * (1 - gate_sigmoid));
        }
    }
}

void swiglu_forward_cuda(torch::Tensor gate, torch::Tensor up, torch::Tensor output, const int BLOCK_SIZE) {
    const int num_elements = gate.numel();

    AT_DISPATCH_CUSTOM_FLOAT_TYPES(
        gate.scalar_type(), "swiglu_forward_cuda_kernel", ([&] {
            const int num_elements_per_thread = get_num_elements_in_vector_dtype<scalar_t, fp32_4>();

            const int num_elements_per_block = BLOCK_SIZE * num_elements_per_thread;
            const int NUM_BLOCKS = (num_elements + num_elements_per_block - 1) / num_elements_per_block;

            _swiglu_forward_cuda_kernel<scalar_t><<<NUM_BLOCKS, BLOCK_SIZE>>>(
                gate.data_ptr<scalar_t>(), up.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(), num_elements);
        }));
}

void swiglu_backward_cuda_kernel(torch::Tensor gate,
                                 torch::Tensor up,
                                 torch::Tensor output_grad,
                                 torch::Tensor gate_grad,
                                 torch::Tensor up_grad,
                                 const int num_elements,
                                 const int BLOCK_SIZE) {
    AT_DISPATCH_CUSTOM_FLOAT_TYPES(
        gate.scalar_type(), "swiglu_backward_cuda_kernel", ([&] {
            const int num_elements_per_thread = get_num_elements_in_vector_dtype<scalar_t, fp32_4>();

            const int num_elements_per_block = BLOCK_SIZE * num_elements_per_thread;
            const int NUM_BLOCKS = (num_elements + num_elements_per_block - 1) / num_elements_per_block;

            _swiglu_backward_cuda_kernel<scalar_t><<<NUM_BLOCKS, BLOCK_SIZE>>>(gate.data_ptr<scalar_t>(),
                                                                               up.data_ptr<scalar_t>(),
                                                                               output_grad.data_ptr<scalar_t>(),
                                                                               gate_grad.data_ptr<scalar_t>(),
                                                                               up_grad.data_ptr<scalar_t>(),
                                                                               num_elements);
        }));
}
