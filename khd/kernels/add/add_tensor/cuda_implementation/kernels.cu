#include "hip/hip_runtime.h"
#include "../../../utils/dtypes.h"
#include "../../../utils/threads.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#define NAIVE_KERNEL_NAME "add_tensor_forward_naive_cuda_kernel"
#define EFFICIENT_KERNEL_NAME "add_tensor_forward_efficient_cuda_kernel"

template <typename scalar_t>
__global__ void _add_tensor_forward_naive_cuda_kernel(const scalar_t *x,
                                                      const scalar_t *y,
                                                      scalar_t *output,
                                                      const int num_elements) {
    const int thread_id = get_global_thread_id();

    if (thread_id < num_elements) {
        output[thread_id] = x[thread_id] + y[thread_id];
    }
}

template <typename scalar_t>
__global__ void _add_tensor_forward_efficient_cuda_kernel(const scalar_t *x,
                                                          const scalar_t *y,
                                                          scalar_t *output,
                                                          const int num_elements) {
    const int thread_id = get_global_thread_id();
    const int num_elements_per_thread = get_num_elements_in_vector_dtype<scalar_t, fp32_4>();

    const int start = thread_id * num_elements_per_thread;
    const int end = (thread_id + 1) * num_elements_per_thread - 1; // inclusive of last element

    using dtype = DType<scalar_t>;
    using T2 = typename dtype::nv_dtype2;

    if (start < num_elements && end < num_elements) {
        const fp32 *x_vec = (fp32 *)&((const fp32_4 *)x)[thread_id];
        const fp32 *y_vec = (fp32 *)&((const fp32_4 *)y)[thread_id];

        fp32 output_buffer[4];

        // clang-format off
        #pragma unroll
        // clang-format on
        for (int i = 0; i < 4; i++) {
            if (std::is_same_v<scalar_t, fp32>) {
                output_buffer[i] = x_vec[i] + y_vec[i];
            } else if constexpr (std::is_same_v<scalar_t, c10::Half> || std::is_same_v<scalar_t, c10::BFloat16>) {
                T2 _x = dtype::reinterpret_32_bits_as_2x16(x_vec[i]);
                T2 _y = dtype::reinterpret_32_bits_as_2x16(y_vec[i]);
                _x = __hadd2(_x, _y);

                output_buffer[i] = dtype::reinterpret_2x16_as_32_bits(_x);
            } else {
                assert(false && "Function not implemented");
            }
        }

        ((fp32_4 *)output)[thread_id] =
            make_float4(output_buffer[0], output_buffer[1], output_buffer[2], output_buffer[3]);
    } else if (start < num_elements) {
        // clang-format off
        #pragma unroll
        // clang-format on
        for (int i = start; i < num_elements; i++) {
            output[i] = x[i] + y[i];
        }
    }
}

void add_tensor_forward_cuda_kernel_dispatch(const torch::Tensor x,
                                             const torch::Tensor y,
                                             torch::Tensor output,
                                             const bool &use_efficient_kernel,
                                             const int &num_elements,
                                             const int &BLOCK_SIZE) {
    str kernel_name;
    if (use_efficient_kernel) {
        kernel_name = EFFICIENT_KERNEL_NAME;
    } else {
        kernel_name = NAIVE_KERNEL_NAME;
    }

    AT_DISPATCH_CUSTOM_FLOAT_TYPES(
        x.scalar_type(), kernel_name, ([&] {
            if (use_efficient_kernel) {
                const int num_elements_per_thread = get_num_elements_in_vector_dtype<scalar_t, fp32_4>();
                const int num_elements_per_block = BLOCK_SIZE * num_elements_per_thread;
                const int NUM_BLOCKS = (num_elements + num_elements_per_block - 1) / num_elements_per_block;

                _add_tensor_forward_efficient_cuda_kernel<scalar_t><<<NUM_BLOCKS, BLOCK_SIZE>>>(
                    x.data_ptr<scalar_t>(), y.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(), num_elements);
            } else {
                const int NUM_BLOCKS = (num_elements + BLOCK_SIZE - 1) / BLOCK_SIZE;

                _add_tensor_forward_naive_cuda_kernel<scalar_t><<<NUM_BLOCKS, BLOCK_SIZE>>>(
                    x.data_ptr<scalar_t>(), y.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(), num_elements);
            }
        }));
}
