#include "hip/hip_runtime.h"
#include "../../../utils/dtypes.h"
#include "../../../utils/threads.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

template <typename scalar_t>
__global__ void _add_tensor_forward_cuda_kernel(const scalar_t *x,
                                                const scalar_t *y,
                                                scalar_t *output,
                                                const int num_elements) {
    const int thread_id = get_global_thread_id();
    const int num_elements_per_thread = get_num_elements_in_vector_dtype<scalar_t, fp32_4>();

    const int start = thread_id * num_elements_per_thread;
    const int end = (thread_id + 1) * num_elements_per_thread - 1; // inclusive of last element

    if (start < num_elements && end < num_elements) {
        const fp32 *x_vec = (fp32 *)&((const fp32_4 *)x)[thread_id];
        const fp32 *y_vec = (fp32 *)&((const fp32_4 *)y)[thread_id];

        fp32 output_buffer[4];

        // clang-format off
        #pragma unroll
        // clang-format on
        for (int i = 0; i < 4; i++) {
            if (std::is_same_v<scalar_t, fp32>) {
                output_buffer[i] = x_vec[i] + y_vec[i];
            } else if constexpr (std::is_same_v<scalar_t, c10::Half> || std::is_same_v<scalar_t, c10::BFloat16>) {
                using dtype = DType<scalar_t>;
                using T2 = typename dtype::nv_dtype2;

                T2 _x = dtype::reinterpret_32_bits_as_2x16(x_vec[i]);
                T2 _y = dtype::reinterpret_32_bits_as_2x16(y_vec[i]);
                _x = __hadd2(_x, _y);

                output_buffer[i] = dtype::reinterpret_2x16_as_32_bits(_x);
            } else {
                assert(false && "Function not implemented");
            }
        }

        ((fp32_4 *)output)[thread_id] =
            make_float4(output_buffer[0], output_buffer[1], output_buffer[2], output_buffer[3]);
    } else if (start < num_elements) {
        // clang-format off
        #pragma unroll
        // clang-format on
        for (int i = start; i < num_elements; i++) {
            output[i] = x[i] + y[i];
        }
    }
}

void add_tensor_forward_cuda_kernel(
    torch::Tensor x, torch::Tensor y, torch::Tensor output, const int num_elements, const int BLOCK_SIZE) {
    AT_DISPATCH_FLOATING_TYPES_AND2(
        at::ScalarType::Half, at::ScalarType::BFloat16, x.scalar_type(), "add_tensor_forward_cuda_kernel", ([&] {
            const int num_elements_per_thread = get_num_elements_in_vector_dtype<scalar_t, fp32_4>();

            const int num_elements_per_block = BLOCK_SIZE * num_elements_per_thread;
            const int NUM_BLOCKS = (num_elements + num_elements_per_block - 1) / num_elements_per_block;

            _add_tensor_forward_cuda_kernel<scalar_t><<<NUM_BLOCKS, BLOCK_SIZE>>>(
                x.data_ptr<scalar_t>(), y.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(), num_elements);
        }));
}
