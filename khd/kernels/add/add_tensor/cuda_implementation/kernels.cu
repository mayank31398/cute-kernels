#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "../../../utils/dtypes.h"
#include "../../../utils/threads.h"

template <typename scalar_t, typename vector_t, int vector_instruction_width>
__global__ void _add_tensor_forward_cuda_kernel(const scalar_t *x,
                                                const scalar_t *y,
                                                scalar_t *output,
                                                const int64_t num_elements) {
    const int64_t thread_id = get_global_thread_id();

    // constexpr avoids error when n == 1 when allocating output_buffer for fp16/bf16
    if constexpr (vector_instruction_width == 1) {
        if (thread_id < num_elements) {
            output[thread_id] = x[thread_id] + y[thread_id];
        }
    } else {
        using dtype = DType<scalar_t>;

        const int64_t start = thread_id * vector_instruction_width;
        const int64_t end = (thread_id + 1) * vector_instruction_width - 1;  // inclusive of last element

        if (start < num_elements && end < num_elements) {
            if constexpr (std::is_same_v<scalar_t, fp32>) {
                const fp32 *_x = (fp32 *)&((vector_t *)x)[thread_id];
                const fp32 *_y = (fp32 *)&((vector_t *)y)[thread_id];
                fp32 output_buffer[vector_instruction_width];

                // clang-format off
                #pragma unroll
                // clang-format on
                for (int i = 0; i < vector_instruction_width; i++) {
                    output_buffer[i] = _x[i] + _y[i];
                }

                if constexpr (std::is_same_v<vector_t, fp32_2>) {
                    assert(vector_instruction_width == 2);
                    ((vector_t *)output)[thread_id] = dtype::make2(output_buffer);
                } else if constexpr (std::is_same_v<vector_t, fp32_4>) {
                    assert(vector_instruction_width == 4);
                    ((vector_t *)output)[thread_id] = dtype::make4(output_buffer);
                }
            } else {
                if constexpr (std::is_same_v<vector_t, fp16_2> || std::is_same_v<vector_t, bf16_2>) {
                    ((vector_t *)output)[thread_id] = __hadd2(((vector_t *)x)[thread_id], ((vector_t *)y)[thread_id]);
                } else {
                    using T = typename dtype::nv_dtype;
                    using T2 = typename dtype::nv_dtype2;

                    const int n = vector_instruction_width / sizeof(T);

                    const fp32 *_x = (fp32 *)&((vector_t *)x)[thread_id];
                    const fp32 *_y = (fp32 *)&((vector_t *)y)[thread_id];

                    fp32 output_buffer[n];

                    // clang-format off
                    #pragma unroll
                    // clang-format on
                    for (int i = 0; i < n; i++) {
                        output_buffer[i] = dtype::reinterpret_2x16_as_32_bits(__hadd2(
                            dtype::reinterpret_32_bits_as_2x16(_x[i]), dtype::reinterpret_32_bits_as_2x16(_y[i])));
                    }

                    if constexpr (std::is_same_v<vector_t, fp32_2>) {
                        assert(vector_instruction_width == 4);
                        ((vector_t *)output)[thread_id] = DType<fp32>::make2(output_buffer);
                    } else if constexpr (std::is_same_v<vector_t, fp32_4>) {
                        assert(vector_instruction_width == 8);
                        ((vector_t *)output)[thread_id] = DType<fp32>::make4(output_buffer);
                    }
                }
            }
        } else if (start < num_elements) {
            // clang-format off
            #pragma unroll
            // clang-format on
            for (int64_t i = start; i < num_elements; i++) {
                output[i] = x[i] + y[i];
            }
        }
    }
}

void add_tensor_forward_cuda(const torch::Tensor x,
                             const torch::Tensor y,
                             const torch::Tensor output,
                             const int &vector_instruction_width,
                             const int &BLOCK_SIZE) {
    const int64_t num_elements = x.numel();

    AT_DISPATCH_CUSTOM_FLOAT_TYPES(
        x.scalar_type(), "add_tensor_forward_cuda_kernel", ([&] {
            const int num_elements_per_block = BLOCK_SIZE * vector_instruction_width;
            const int NUM_BLOCKS = (num_elements + num_elements_per_block - 1) / num_elements_per_block;

            switch (vector_instruction_width) {
                case 1:
                    _add_tensor_forward_cuda_kernel<scalar_t, scalar_t, 1><<<NUM_BLOCKS, BLOCK_SIZE>>>(
                        x.data_ptr<scalar_t>(), y.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(), num_elements);
                    break;
                case 2:
                    using vector_t = typename DType<scalar_t>::nv_dtype2;
                    _add_tensor_forward_cuda_kernel<scalar_t, vector_t, 2><<<NUM_BLOCKS, BLOCK_SIZE>>>(
                        x.data_ptr<scalar_t>(), y.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(), num_elements);
                    break;
                case 4:
                    if constexpr (std::is_same_v<scalar_t, fp32>) {
                        _add_tensor_forward_cuda_kernel<scalar_t, fp32_4, 4><<<NUM_BLOCKS, BLOCK_SIZE>>>(
                            x.data_ptr<scalar_t>(), y.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(), num_elements);
                    } else {
                        _add_tensor_forward_cuda_kernel<scalar_t, fp32_2, 4><<<NUM_BLOCKS, BLOCK_SIZE>>>(
                            x.data_ptr<scalar_t>(), y.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(), num_elements);
                    }
                    break;
                case 8:
                    if constexpr (std::is_same_v<scalar_t, fp32>) {
                        assert(false);
                    } else {
                        _add_tensor_forward_cuda_kernel<scalar_t, fp32_4, 8><<<NUM_BLOCKS, BLOCK_SIZE>>>(
                            x.data_ptr<scalar_t>(), y.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(), num_elements);
                    }
                    break;
                default:
                    throw std::runtime_error("invalid vector_instruction_width");
                    break;
            }
        }));
}
