#include "hip/hip_runtime.h"
#include "../../../utils/dtypes.h"
#include "../../../utils/threads.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

template <typename scalar_t, typename vector_t>
__global__ void _add_tensor_forward_cuda_kernel(const scalar_t *x,
                                                const scalar_t *y,
                                                scalar_t *output,
                                                const int num_elements,
                                                const int vectorized_load_store_size) {
    const int thread_id = get_global_thread_id();

    if (vectorized_load_store_size == 1) {
        if (thread_id < num_elements) {
            output[thread_id] = x[thread_id] + y[thread_id];
        }
    } else {
        using dtype = DType<scalar_t>;

        const int start = thread_id * vectorized_load_store_size;
        const int end = (thread_id + 1) * vectorized_load_store_size - 1; // inclusive of last element

        if (start < num_elements && end < num_elements) {
            using T = typename dtype::nv_dtype;

            if constexpr (std::is_same_v<scalar_t, fp32>) {
                const T *_x = (T *)&((vector_t *)x)[thread_id];
                const T *_y = (T *)&((vector_t *)y)[thread_id];
                T *output_buffer = new T[vectorized_load_store_size];

                // clang-format off
                #pragma unroll
                // clang-format on
                for (int i = 0; i < vectorized_load_store_size; i++) {
                    output_buffer[i] = _x[i] + _y[i];
                }

                if constexpr (std::is_same_v<vector_t, fp32_2>) {
                    assert(vectorized_load_store_size == 2);
                    ((vector_t *)output)[thread_id] = dtype::make2(output_buffer);
                } else if constexpr (std::is_same_v<vector_t, fp32_4>) {
                    assert(vectorized_load_store_size == 4);
                    ((vector_t *)output)[thread_id] = dtype::make4(output_buffer);
                }
            } else {
                if constexpr (std::is_same_v<vector_t, fp16_2> || std::is_same_v<vector_t, bf16_2>) {
                    ((vector_t *)output)[thread_id] = __hadd2(((vector_t *)x)[thread_id], ((vector_t *)y)[thread_id]);
                } else {
                    using T2 = typename dtype::nv_dtype2;

                    fp32 *output_buffer = new fp32[2 * vectorized_load_store_size];

                    // clang-format off
                    #pragma unroll
                    // clang-format on
                    for (int i = 0; i < 2 * vectorized_load_store_size; i += 2) {
                        T2 _x = dtype::reinterpret_32_bits_as_2x16(((vector_t *)x)[i]);
                        T2 _y = dtype::reinterpret_32_bits_as_2x16(((vector_t *)y)[i]);

                        output_buffer[i] = dtype::reinterpret_2x16_as_32_bits(__hadd2(_x, _y));
                    }

                    if constexpr (std::is_same_v<vector_t, fp32_2>) {
                        assert(vectorized_load_store_size == 4);
                        ((vector_t *)output)[thread_id] = dtype::make2(output_buffer);
                    } else if constexpr (std::is_same_v<vector_t, fp32_4>) {
                        assert(vectorized_load_store_size == 8);
                        ((vector_t *)output)[thread_id] = dtype::make4(output_buffer);
                    }
                }
            }
        } else if (start < num_elements) {
            // clang-format off
            #pragma unroll
            // clang-format on
            for (int i = start; i < num_elements; i++) {
                output[i] = x[i] + y[i];
            }
        }
    }
}

void add_tensor_forward_cuda_kernel_dispatch(const torch::Tensor x,
                                             const torch::Tensor y,
                                             const torch::Tensor output,
                                             const int &vectorized_load_store_size,
                                             const int &num_elements,
                                             const int &BLOCK_SIZE) {
    AT_DISPATCH_CUSTOM_FLOAT_TYPES(x.scalar_type(), "add_tensor_forward_cuda_kernel", ([&] {
                                       const int num_elements_per_block = BLOCK_SIZE * vectorized_load_store_size;
                                       const int NUM_BLOCKS =
                                           (num_elements + num_elements_per_block - 1) / num_elements_per_block;

                                       switch (vectorized_load_store_size) {
                                       case 1:
                                           _add_tensor_forward_cuda_kernel<scalar_t, scalar_t>
                                               <<<NUM_BLOCKS, BLOCK_SIZE>>>(x.data_ptr<scalar_t>(),
                                                                            y.data_ptr<scalar_t>(),
                                                                            output.data_ptr<scalar_t>(),
                                                                            num_elements,
                                                                            vectorized_load_store_size);
                                           break;
                                       case 2:
                                           using vector_t = typename DType<scalar_t>::nv_dtype2;
                                           _add_tensor_forward_cuda_kernel<scalar_t, vector_t>
                                               <<<NUM_BLOCKS, BLOCK_SIZE>>>(x.data_ptr<scalar_t>(),
                                                                            y.data_ptr<scalar_t>(),
                                                                            output.data_ptr<scalar_t>(),
                                                                            num_elements,
                                                                            vectorized_load_store_size);
                                           break;
                                       default:
                                           throw std::runtime_error("invalid vectorized_load_store_size");
                                           break;
                                       }

                                       // }
                                       // case 4:
                                       //     if constexpr (std::is_same_v<scalar_t, fp32>) {
                                       //         _add_tensor_forward_cuda_kernel<fp32, fp32_4>
                                       //             <<<NUM_BLOCKS, BLOCK_SIZE>>>(x.data_ptr<scalar_t>(),
                                       //                                          y.data_ptr<scalar_t>(),
                                       //                                          output.data_ptr<scalar_t>(),
                                       //                                          num_elements,
                                       //                                          vectorized_load_store_size);
                                       //     } else {
                                       //         _add_tensor_forward_cuda_kernel<scalar_t, fp32_2>
                                       //             <<<NUM_BLOCKS, BLOCK_SIZE>>>(x.data_ptr<scalar_t>(),
                                       //                                          y.data_ptr<scalar_t>(),
                                       //                                          output.data_ptr<scalar_t>(),
                                       //                                          num_elements,
                                       //                                          vectorized_load_store_size);
                                       //     }
                                       //     break;
                                       // default:
                                       //     _add_tensor_forward_cuda_kernel<scalar_t, fp32_4>
                                       //         <<<NUM_BLOCKS, BLOCK_SIZE>>>(x.data_ptr<scalar_t>(),
                                       //                                      y.data_ptr<scalar_t>(),
                                       //                                      output.data_ptr<scalar_t>(),
                                       //                                      num_elements,
                                       //                                      vectorized_load_store_size);
                                       //     break;
                                       // }

                                       // if (!kernel_func) {
                                       //     throw std::runtime_error("Kernel function is not set correctly");
                                       // }

                                       // hipError_t err = hipGetLastError();
                                       // if (err != hipSuccess) {
                                       //     throw std::runtime_error("Kernel launch failed: " +
                                       //     std::string(hipGetErrorString(err)));
                                       // }
                                   }));
}
