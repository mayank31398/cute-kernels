#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "../../../utils/dtypes.h"
#include "../../../utils/threads.h"

template <typename scalar_t, typename vector_t>
__global__ void _add_tensor_forward_cuda_kernel(const scalar_t *x,
                                                const scalar_t *y,
                                                scalar_t *output,
                                                const int64_t num_elements) {
    const int vector_instruction_width = sizeof(vector_t) / sizeof(scalar_t);
    const int64_t thread_id = get_global_thread_id();

    // no vector instructions
    if constexpr (vector_instruction_width == 1) {
        if (thread_id < num_elements) {
            output[thread_id] = x[thread_id] + y[thread_id];
        }
    } else {
        // no vector instructions
        using dtype = DType<scalar_t>;

        const int64_t start = thread_id * vector_instruction_width;
        const int64_t end = (thread_id + 1) * vector_instruction_width - 1;  // inclusive of last element

        if (start < num_elements && end < num_elements) {
            vector_t *output_vec = (vector_t *)output;

            if constexpr (std::is_same_v<scalar_t, fp32>) {
                const fp32 *x_vec = (fp32 *)&((vector_t *)x)[thread_id];
                const fp32 *y_vec = (fp32 *)&((vector_t *)y)[thread_id];
                fp32 output_buffer[vector_instruction_width];

                // clang-format off
                #pragma unroll
                // clang-format on
                for (int i = 0; i < vector_instruction_width; i++) {
                    output_buffer[i] = x_vec[i] + y_vec[i];
                }

                if constexpr (std::is_same_v<vector_t, fp32_2>) {
                    static_assert(vector_instruction_width == 2);
                    output_vec[thread_id] = dtype::make2(output_buffer);
                } else if constexpr (std::is_same_v<vector_t, fp32_4>) {
                    static_assert(vector_instruction_width == 4);
                    output_vec[thread_id] = dtype::make4(output_buffer);
                }
            } else {
                using T2 = DType<scalar_t>;

                if constexpr (std::is_same_v<vector_t, fp16_2> || std::is_same_v<vector_t, bf16_2>) {
                    T2 _x = ((vector_t *)x)[thread_id];
                    T2 _y = ((vector_t *)y)[thread_id];

                    output_vec[thread_id] = __hadd2(_x, _y);
                } else {
                    const fp32 *x_vec = (fp32 *)&((vector_t *)x)[thread_id];
                    const fp32 *y_vec = (fp32 *)&((vector_t *)y)[thread_id];

                    const int n = vector_instruction_width >> 1;
                    fp32 output_buffer[n];

                    // clang-format off
                    #pragma unroll
                    // clang-format on
                    for (int i = 0; i < n; i++) {
                        T2 _x = dtype::reinterpret_32_bits_as_2x16(x_vec[i]);
                        T2 _y = dtype::reinterpret_32_bits_as_2x16(y_vec[i]);

                        _x = __hadd2(_x, _y);
                        output_buffer[i] = dtype::reinterpret_2x16_as_32_bits(_x);
                    }

                    if constexpr (std::is_same_v<vector_t, fp32_2>) {
                        assert(vector_instruction_width == 4);
                        output_vec[thread_id] = DType<fp32>::make2(output_buffer);
                    } else if constexpr (std::is_same_v<vector_t, fp32_4>) {
                        assert(vector_instruction_width == 8);
                        output_vec[thread_id] = DType<fp32>::make4(output_buffer);
                    }
                }
            }
        } else if (start < num_elements) {
            // clang-format off
            #pragma unroll
            // clang-format on
            for (int64_t i = start; i < num_elements; i++) {
                output[i] = x[i] + y[i];
            }
        }
    }
}

void add_tensor_forward_cuda(const torch::Tensor x,
                             const torch::Tensor y,
                             torch::Tensor output,
                             const int &vector_instruction_width,
                             const int &BLOCK_SIZE) {
    const int64_t num_elements = x.numel();

    AT_DISPATCH_CUSTOM_FLOAT_TYPES(
        x.scalar_type(), "add_tensor_forward_cuda_kernel", ([&] {
            const int num_elements_per_block = BLOCK_SIZE * vector_instruction_width;
            const int NUM_BLOCKS = (num_elements + num_elements_per_block - 1) / num_elements_per_block;

            switch (vector_instruction_width) {
                case 1:
                    _add_tensor_forward_cuda_kernel<scalar_t, scalar_t><<<NUM_BLOCKS, BLOCK_SIZE>>>(
                        x.data_ptr<scalar_t>(), y.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(), num_elements);
                    break;
                case 2:
                    using vector_t = typename DType<scalar_t>::nv_dtype2;
                    _add_tensor_forward_cuda_kernel<scalar_t, vector_t><<<NUM_BLOCKS, BLOCK_SIZE>>>(
                        x.data_ptr<scalar_t>(), y.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(), num_elements);
                    break;
                case 4:
                    if constexpr (std::is_same_v<scalar_t, fp32>) {
                        _add_tensor_forward_cuda_kernel<scalar_t, fp32_4><<<NUM_BLOCKS, BLOCK_SIZE>>>(
                            x.data_ptr<scalar_t>(), y.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(), num_elements);
                    } else {
                        _add_tensor_forward_cuda_kernel<scalar_t, fp32_2><<<NUM_BLOCKS, BLOCK_SIZE>>>(
                            x.data_ptr<scalar_t>(), y.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(), num_elements);
                    }
                    break;
                case 8:
                    if constexpr (std::is_same_v<scalar_t, fp32>) {
                        throw std::runtime_error("fp32 doesn't support vector_instruction_width = 8");
                    } else {
                        _add_tensor_forward_cuda_kernel<scalar_t, fp32_4><<<NUM_BLOCKS, BLOCK_SIZE>>>(
                            x.data_ptr<scalar_t>(), y.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(), num_elements);
                    }
                    break;
                default:
                    throw std::runtime_error("invalid vector_instruction_width");
                    break;
            }
        }));
}
